#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include <cmath>

#include "particleSystem.h"
#include "saiga/core/util/assert.h"

//#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/device_helper.h"

#include "saiga/cuda/memory.h"

// 4.0
#include <Eigen/Core>
#include "saiga/core/math/random.h"

#include "svd3_cuda.h"

// 4.4
#include "saiga/core/geometry/AccelerationStructure.h"
#include "saiga/core/geometry/intersection.h"

void ParticleSystem::setDevicePtr(void* particleVbo) {
    d_particles = ArrayView<Particle>((Particle*) particleVbo, particleCount);
}

__global__ void updateParticlesPBD1_radius(float dt, vec3 gravity, Saiga::ArrayView<Particle>particles, float dampV, float particleRadiusWater, float particleRadiusCloth) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (p.fixed)
        return;

    if (p.rbID == -2)
        p.radius = particleRadiusWater;
    else if (p.rbID == -3)
        p.radius = particleRadiusCloth;

    /*
    // p.velocity += dt * p.massinv * gravity; // falsch auf folie
    p.velocity += dt * gravity;
    // dampVelocities
    p.velocity *= dampV;

    p.predicted = p.position + dt * p.velocity;*/
    vec3 newVelocity = p.velocity + dt * gravity;
    // dampVelocities
    newVelocity *= dampV;

    p.predicted = p.position + dt * newVelocity;
}

__global__ void updateParticlesPBD2Iterator(float dt, Saiga::ArrayView<Particle>particles, float relaxP) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (!p.fixed) {
        p.predicted += relaxP * p.d_predicted;
    }
    // reset
    p.d_predicted = {0, 0, 0};
}
__global__ void updateParticlesPBD2(float dt, Saiga::ArrayView<Particle>particles, float relaxP) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (!p.fixed) {
        //p.predicted += relaxP * p.d_predicted;
        p.velocity = (p.predicted - p.position) / dt;
        p.position = p.predicted;
    }
    // reset
    p.d_predicted = {0, 0, 0};

    // 6.2
    p.lambda = 0;
}

__global__ void resetParticles(int x, int z, vec3 corner, float distance, Saiga::ArrayView<Particle>particles, float randInitMul, float particleRenderRadius, int rbID, vec4 color) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    int xPos = (ti.thread_id) % x;
    int zPos = ((ti.thread_id - xPos) / x) % z;
    int yPos = (((ti.thread_id - xPos) / x) - zPos) / z;
    vec3 pos = {xPos, yPos, zPos};

    // pseudo random position offset
    int rand = ti.thread_id + p.position[0];
    p.position = corner + pos * distance + vec3{rand % 11, rand % 17, rand % 13} * randInitMul;

    p.velocity = {0, 0, 0};
    p.massinv = 1.0/1.0;
    p.predicted = p.position;
    // 2.3
    p.color = color;
    p.radius = particleRenderRadius;

    p.fixed = false;

    // 4.0
    p.rbID = rbID;
    p.relative = {0,0,0};
    p.sdf = {0,0,0};

    // 6.0
    p.lambda = 0;

    p.id = ti.thread_id; // cloth
}

__global__ void initParticles(int startIdx, int count, int x, int z, vec3 corner, float distance, Saiga::ArrayView<Particle>particles, float randInitMul, float particleRenderRadius, int rbID, vec4 color, bool fixed=false, float mass=1.0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    if (ti.thread_id < startIdx || ti.thread_id >= startIdx + count)
        return;

    int idx = ti.thread_id - startIdx;
    Particle &p = particles[ti.thread_id];

    int xPos = (idx) % x;
    int zPos = ((idx - xPos) / x) % z;
    int yPos = (((idx - xPos) / x) - zPos) / z;
    vec3 pos = {xPos, yPos, zPos};

    // pseudo random position offset
    int rand = ti.thread_id + p.position[0];
    p.position = corner + pos * distance + vec3{rand % 11, rand % 17, rand % 13} * randInitMul;

    p.velocity = {0, 0, 0};
    p.massinv = 1.0f/mass;
    p.predicted = p.position;
    // 2.3
    p.color = color;
    p.radius = particleRenderRadius;

    p.fixed = fixed;

    // 4.0
    p.rbID = rbID;
    p.relative = {0,0,0};
    p.sdf = {0,0,0};

    // 6.0
    p.lambda = 0;
}

// 4.0 TODO fix or remove
__global__ void initCuboidParticles(Saiga::ArrayView<Particle> particles, int id, vec3 pos, ivec3 dim, vec3 rot, vec4 color, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id > 0)
        return;
    
    mat3 rotMat;
    rotMat = Eigen::AngleAxisf(rot.x(), vec3::UnitZ())
        * Eigen::AngleAxisf(rot.y(), vec3::UnitY())
        * Eigen::AngleAxisf(rot.z(), vec3::UnitZ());
    
    int count = dim.x() * dim.y() * dim.z();

    for (int i = 0; i < dim.x(); i++) {
        for (int j = 0; j < dim.y(); j++) {
            for (int k = 0; k < dim.z(); k++) {
                vec3 p = {i, j, k};
                p = rotMat * p;
                p += pos;
                particles[particleCountRB].position = p;
                particles[particleCountRB].predicted = p;
                particles[particleCountRB].rbID = id;

                particles[particleCountRB].color = color;

                //4.4
                ivec3 idx = {i, j, k};
                ivec3 dim2 = (dim/2);
                ivec3 dir;
                dir.x() = idx.x() < dim2.x() ? idx.x() - dim2.x() : dim2.x() - (dim.x() - idx.x() - 1);
                dir.y() = idx.y() < dim2.y() ? idx.y() - dim2.y() : dim2.y() - (dim.y() - idx.y() - 1);
                dir.z() = idx.z() < dim2.z() ? idx.z() - dim2.z() : dim2.z() - (dim.z() - idx.z() - 1);
                //float m = min(min(fabs(sdf.x()), fabs(sdf.y())), fabs(sdf.z()));

                ivec3 absdir = {abs(dir.x()), abs(dir.y()), abs(dir.z())};

                int minDir = max(max(absdir.x(), absdir.y()), absdir.z());
                vec3 sdf = {0,0,0};

                ivec3 dirSign = dir;
                dirSign.x() = dirSign.x() > 0 ? 1 : dirSign.x();
                dirSign.x() = dirSign.x() < 0 ? -1 : dirSign.x();
                dirSign.y() = dirSign.y() > 0 ? 1 : dirSign.y();
                dirSign.y() = dirSign.y() < 0 ? -1 : dirSign.y();
                dirSign.z() = dirSign.z() > 0 ? 1 : dirSign.z();
                dirSign.z() = dirSign.z() < 0 ? -1 : dirSign.z();

                if (absdir.x() == minDir)
                    sdf.x() = dirSign.x();
                if (absdir.y() == minDir)
                    sdf.y() = dirSign.y();
                if (absdir.z() == minDir)
                    sdf.z() = dirSign.z();


                int mx = min(i + 1, dim.x() - i);
                int my = min(j + 1, dim.y() - j);
                int mz = min(k + 1, dim.z() - k);

                float m = min(min(mx, my), mz);

                particles[particleCountRB].sdf = -m * normalize(sdf);// minus to point inwards

                printf("%i %i %i, %f, %f, %f, %f\n", i, j, k, sdf.x(), sdf.y(), sdf.z(), m);

                particleCountRB++;
            }
        }
    }

    rigidBodies[id].particleCount = count;
}

__global__ void initSingleRigidBodyParticle(Saiga::ArrayView<Particle> particles, int id, vec3 pos, vec3 sdf, vec4 color, int particleCountRB, RigidBody *rigidBodies, bool fixed=false, float mass=1.0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id > 0)
        return;
    
    particles[particleCountRB].position = pos;
    particles[particleCountRB].predicted = pos;
    particles[particleCountRB].rbID = id;

    particles[particleCountRB].color = color;

    particles[particleCountRB].fixed = fixed;
    particles[particleCountRB].massinv = 1.0f/mass;

    // 4.4
    particles[particleCountRB].sdf = sdf;

    rigidBodies[id].particleCount++;
}

__global__ void initRigidBodyParticles(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    
    particles[ti.thread_id].relative = particles[ti.thread_id].predicted - rigidBodies[particles[ti.thread_id].rbID].originOfMass;
}

// 4.4
int ParticleSystem::loadObj(int rigidBodyCount, int particleCountRB, vec3 pos, vec3 rot, vec4 color) {
    Saiga::UnifiedModel model("objs/teapot.obj");
    Saiga::UnifiedMesh mesh = model.CombinedMesh().first;
    std::vector<Triangle> triangles = mesh.TriangleSoup();
    // 1
    Saiga::AABB bb = model.BoundingBox(); // mesh. or model.BoundingBox()
    vec3 min = bb.min;
    vec3 max = bb.max;
    // 2
    // Schnittstellen
    float maxObjParticleCount = 40;
    float maxSize = bb.maxSize();
    //float sampleDistance = 0.1;
    float sampleDistance = maxSize / maxObjParticleCount;
    int count = 0;
    Saiga::AccelerationStructure::ObjectMedianBVH omBVH(triangles);

    if (true) {
        // 3d voxel grid
        vec3 size = bb.Size() / sampleDistance;
        const int xDim = ceil(size.x());
        const int yDim = ceil(size.y());
        const int zDim = ceil(size.z());

        auto ***voxel = new std::pair<int, vec3>**[zDim];
        for(int i = 0; i < zDim; ++i) {
            voxel[i] = new std::pair<int, vec3>*[yDim];
            for(int j = 0; j < yDim; ++j) {
                voxel[i][j] = new std::pair<int, vec3>[xDim];
            }
        }
        // init voxels
        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    bool isInside = true;
                    for (float dx = -1; dx < 2; dx += 2) {
                        for (float dy = -1; dy < 2; dy += 2) {
                            for (float dz = -1; dz < 2; dz += 2) {
                                vec3 dir = {dx,dy,dz};
                                Saiga::Ray ray(dir, ori);
                                Saiga::Intersection::RayTriangleIntersection rti = omBVH.getClosest(ray);
                                if (!rti.valid)
                                    isInside = false;
                            }
                        }
                    }
                    if (isInside) {
                        count++;
                        voxel[z][y][x].first = 1;
                    } else {
                        voxel[z][y][x].first = 0;
                    }
                    // init border sdf
                    voxel[z][y][x].second = {0,0,0};
                    if (x == 0)
                        voxel[z][y][x].second[0] = +1;
                    else if (x == xDim-1)
                        voxel[z][y][x].second[0] = -1;
                    
                    if (y == 0)
                        voxel[z][y][x].second[1] = +1;
                    else if (y == yDim-1)
                        voxel[z][y][x].second[1] = -1;
                    
                    if (z == 0)
                        voxel[z][y][x].second[2] = +1;
                    else if (z == zDim-1)
                        voxel[z][y][x].second[2] = -1;
                }
            }
        }
        // calc distance field
        int i = 0;
        int changed = 1;
        while (changed) {
            i++;
            changed = 0;
            for (int x = 1; x < xDim-1; x++) {
                for (int y = 1; y < yDim-1; y++) {
                    for (int z = 1; z < zDim-1; z++) {
                        if (voxel[z][y][x].first == i) {
                            if (    voxel[z+1][y][x].first < i
                                ||  voxel[z-1][y][x].first < i
                                ||  voxel[z][y+1][x].first < i
                                ||  voxel[z][y-1][x].first < i
                                ||  voxel[z][y][x+1].first < i
                                ||  voxel[z][y][x-1].first < i)
                                continue;
                            voxel[z][y][x].first++;
                            changed++;
                        }
                    }
                }
            }
        }
        // calc derivative (normal)
        for (int x = 1; x < xDim-1; x++) {
            for (int y = 1; y < yDim-1; y++) {
                for (int z = 1; z < zDim-1; z++) {
                    if (voxel[z][y][x].first) {
                        float dz = voxel[z+1][y][x].first - voxel[z-1][y][x].first;
                        float dy = voxel[z][y+1][x].first - voxel[z][y-1][x].first;
                        float dx = voxel[z][y][x+1].first - voxel[z][y][x-1].first;
                        voxel[z][y][x].second = {dx, dy, dz};
                    }
                }
            }
        }

        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    if (voxel[z][y][x].first) {
                        count++;
                        float scaling = 1.0f;
                        vec3 position = pos + ori*(scaling / sampleDistance);
                        vec3 sdf = (float)voxel[z][y][x].first * normalize(voxel[z][y][x].second);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, sdf, color, particleCountRB++, d_rigidBodies);
                    }
                }
            }
        }


        for(int i = 0; i < zDim; ++i) {
            for(int j = 0; j < yDim; ++j) {
                delete [] voxel[i][j];
            }
            delete [] voxel[i];
        }
        delete [] voxel;

    } else {
        
        for (float x = min.x(); x < max.x(); x += sampleDistance) {
            for (float y = min.y(); y < max.y(); y += sampleDistance) {
                for (float z = min.z(); z < max.z(); z += sampleDistance) {
                    vec3 ori = {x,y,z};
                    bool isInside = true;
                    for (float dx = -1; dx < 2; dx += 2) {
                        for (float dy = -1; dy < 2; dy += 2) {
                            for (float dz = -1; dz < 2; dz += 2) {
                                vec3 dir = {dx,dy,dz};
                                Saiga::Ray ray(dir, ori);
                                Saiga::Intersection::RayTriangleIntersection rti = omBVH.getClosest(ray);
                                if (!rti.valid)
                                    isInside = false;
                            }
                        }
                    }
                    if (isInside) {
                        count++;
                        float scaling = 1.0f;
                        vec3 position = pos + ori * (scaling / sampleDistance);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, vec3{0.f,0.f,0.f}, color, particleCountRB++, d_rigidBodies);
                    }
                }
            }
        }

    }
    return count;
}

// 4.4
int ParticleSystem::loadBox(int rigidBodyCount, int particleCountRB, ivec3 dim, vec3 pos, vec3 rot, vec4 color, bool fixed=false, float mass=1.0) {    
    vec3 min = {0,0,0};
    int count = 0;
    float sampleDistance = 1.0;

        // 3d voxel grid
        //vec3 size = bb.Size() / sampleDistance;
        const int xDim = dim.x();
        const int yDim = dim.y();
        const int zDim = dim.z();

        auto ***voxel = new std::pair<int, vec3>**[zDim];
        for(int i = 0; i < zDim; ++i) {
            voxel[i] = new std::pair<int, vec3>*[yDim];
            for(int j = 0; j < yDim; ++j) {
                voxel[i][j] = new std::pair<int, vec3>[xDim];
            }
        }
        // init voxels
        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    
                    voxel[z][y][x].first = 1;

                    // init border sdf
                    voxel[z][y][x].second = {0,0,0};
                    if (x == 0)
                        voxel[z][y][x].second[0] = +1;
                    else if (x == xDim-1)
                        voxel[z][y][x].second[0] = -1;
                    
                    if (y == 0)
                        voxel[z][y][x].second[1] = +1;
                    else if (y == yDim-1)
                        voxel[z][y][x].second[1] = -1;
                    
                    if (z == 0)
                        voxel[z][y][x].second[2] = +1;
                    else if (z == zDim-1)
                        voxel[z][y][x].second[2] = -1;
                }
            }
        }
        // calc distance field
        int i = 0;
        int changed = 1;
        while (changed) {
            i++;
            changed = 0;
            for (int x = 1; x < xDim-1; x++) {
                for (int y = 1; y < yDim-1; y++) {
                    for (int z = 1; z < zDim-1; z++) {
                        if (voxel[z][y][x].first == i) {
                            if (    voxel[z+1][y][x].first < i
                                ||  voxel[z-1][y][x].first < i
                                ||  voxel[z][y+1][x].first < i
                                ||  voxel[z][y-1][x].first < i
                                ||  voxel[z][y][x+1].first < i
                                ||  voxel[z][y][x-1].first < i)
                                continue;
                            voxel[z][y][x].first++;
                            changed++;
                        }
                    }
                }
            }
        }
        // calc derivative (normal)
        for (int x = 1; x < xDim-1; x++) {
            for (int y = 1; y < yDim-1; y++) {
                for (int z = 1; z < zDim-1; z++) {
                    if (voxel[z][y][x].first) {
                        float dz = voxel[z+1][y][x].first - voxel[z-1][y][x].first;
                        float dy = voxel[z][y+1][x].first - voxel[z][y-1][x].first;
                        float dx = voxel[z][y][x+1].first - voxel[z][y][x-1].first;
                        voxel[z][y][x].second = {dx, dy, dz};
                    }
                }
            }
        }

        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    if (voxel[z][y][x].first) {
                        count++;
                        float scaling = 1.0f;
                        vec3 position = pos + ori*(scaling / sampleDistance);
                        vec3 sdf = (float)voxel[z][y][x].first * normalize(voxel[z][y][x].second);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, sdf, color, particleCountRB++, d_rigidBodies, fixed, mass);
                    }
                }
            }
        }


        for(int i = 0; i < zDim; ++i) {
            for(int j = 0; j < yDim; ++j) {
                delete [] voxel[i][j];
            }
            delete [] voxel[i];
        }
        delete [] voxel;

    return count;
}

__global__ void caclulateRigidBodyOriginOfMass(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        vec3 d_originOfMass = p.predicted / (float)rigidBodies[p.rbID].particleCount;
        atomicAdd(&rigidBodies[p.rbID].originOfMass[0], d_originOfMass[0]);
        atomicAdd(&rigidBodies[p.rbID].originOfMass[1], d_originOfMass[1]);
        atomicAdd(&rigidBodies[p.rbID].originOfMass[2], d_originOfMass[2]);
    }
}

__global__ void covariance(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        //vec3 pc = p.position - rigidBodies[p.rbID].originOfMass;
        mat3 pcr = (p.predicted - rigidBodies[p.rbID].originOfMass) * p.relative.transpose();

        atomicAdd(&rigidBodies[p.rbID].A(0,0), pcr(0,0));
        atomicAdd(&rigidBodies[p.rbID].A(0,1), pcr(0,1));
        atomicAdd(&rigidBodies[p.rbID].A(0,2), pcr(0,2));
        atomicAdd(&rigidBodies[p.rbID].A(1,0), pcr(1,0));
        atomicAdd(&rigidBodies[p.rbID].A(1,1), pcr(1,1));
        atomicAdd(&rigidBodies[p.rbID].A(1,2), pcr(1,2));
        atomicAdd(&rigidBodies[p.rbID].A(2,0), pcr(2,0));
        atomicAdd(&rigidBodies[p.rbID].A(2,1), pcr(2,1));
        atomicAdd(&rigidBodies[p.rbID].A(2,2), pcr(2,2));
    }
}

__global__ void SVD(RigidBody *rigidBodies, int rigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= rigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    rb.A = svd3_cuda::pd(rb.A);
}

__global__ void resolveRigidBodyConstraints(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        // dx = (Q*r + c) - p
        p.predicted += (rigidBodies[p.rbID].A * p.relative + rigidBodies[p.rbID].originOfMass) - p.predicted;
    }
}

__global__ void resetRigidBody(RigidBody *rigidBodies, int rigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= rigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    // reset
    rb.originOfMass = {0,0,0};
    rb.A = mat3::Zero().cast<float>();
}

__global__ void resetRigidBodyComplete(RigidBody *rigidBodies, int maxRigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= maxRigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    // reset
    rb.particleCount = 0;
    rb.originOfMass = {0,0,0};
    rb.A = mat3::Zero().cast<float>();
}

void ParticleSystem::constraintsShapeMatchingRB() {
    updateRigidBodies();

    resolveRigidBodyConstraints<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();    
}

void ParticleSystem::updateRigidBodies() {
    const unsigned int BLOCKS_RB = Saiga::CUDA::getBlockCount(rigidBodyCount, BLOCK_SIZE);

    resetRigidBody<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();

    caclulateRigidBodyOriginOfMass<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    covariance<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    SVD<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();
    
}

// sehr haesslich
__global__ void deactivateNonRB(Saiga::ArrayView<Particle> particles) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    
    Particle &p = particles[ti.thread_id];
    if (p.rbID < 0) {
        p.position[1] += 1000000.0f;
        p.predicted[1] = p.position[1];
    }
}

void ParticleSystem::reset(int x, int z, vec3 corner, float distance, float randInitMul, int scenario) {
    int rbID = -1; // free particles
    vec4 color = {0.0f, 1.0f, 0.0f, 1.f};
    if (scenario >= 7) {
        color = {0.1f, 0.2f, 0.8f, 1.f};
        rbID = -2; // fluid
    }
    resetParticles<<<BLOCKS, BLOCK_SIZE>>>(x, z, corner, distance, d_particles, randInitMul, particleRenderRadius, rbID, color);
    CUDA_SYNC_CHECK_ERROR();

    if (scenario == 9) {
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(0, 20*20*20, 20, 20, {-20, 0, -20}, distance, d_particles, randInitMul, particleRenderRadius, -2, {0.f, 0.f, 1.f, .1f}, false);
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(20*20*20, 40*40*40, 40, 40, {0, 0, 0}, distance, d_particles, randInitMul, particleRenderRadius, -2, {0.f, .2f, .7f, .3f}, false);
        CUDA_SYNC_CHECK_ERROR();
    }
    
    const unsigned int BLOCKS_RB = Saiga::CUDA::getBlockCount(maxRigidBodyCount, BLOCK_SIZE);
    resetRigidBodyComplete<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, maxRigidBodyCount);

    particleCountRB = 0;
    rigidBodyCount = 0;

    if (scenario == 7) {
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000, 10*5*10, 10, 5, {-2, 0, -2}, 1, d_particles, randInitMul, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000+10*5*10, 64, 4, 4, {0, 0, 10}, 1, d_particles, randInitMul, particleRenderRadius, -1, {.0f, .9f, .0f, 1.f}, false);
        CUDA_SYNC_CHECK_ERROR();
    }

    if (scenario == 8) {
        //initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000, 19*2*10, 2, 19, {-1, 0, -20}, 1, d_particles, 0, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
        //initParticles<<<BLOCKS, BLOCK_SIZE>>>(11000, 19*2*10, 2, 19, {-1, 0, 1.5}, 1, d_particles, 0, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
    }

    if (scenario == 10) { // cloth
        rbID = -3; // free particles
        vec4 color = {1.0f, 1.0f, 1.0f, 1.f};
        resetParticles<<<BLOCKS, BLOCK_SIZE>>>(x, z, corner, distance, d_particles, randInitMul, particleRenderRadius, rbID, color);
        CUDA_SYNC_CHECK_ERROR();


        std::vector<ClothConstraint> clothConstraints(0);

        std::vector<ClothBendingConstraint> clothBendingConstraints(0);

        int dimX = 50;
        int dimZ = 50;

        for (int j = 0; j < dimZ; j++) {
            for (int i = 0; i < dimX; i++) {
                int idx = j * dimX + i;
                if (i < dimX - 1) {
                    clothConstraints.push_back({idx, idx+1, 1.0f * distance});
                }
                if (j < dimZ - 1) {
                    clothConstraints.push_back({idx, idx+dimX, 1.0f * distance});
                }
                if (j < dimZ - 1 && i < dimX - 1) {
                    if (i+j % 2)
                        clothConstraints.push_back({idx, idx+dimX+1, 1.4142f*distance});
                    else
                        clothConstraints.push_back({idx+dimX, idx+1, 1.4142f*distance});

                    clothBendingConstraints.push_back({idx+dimX+1, idx, idx+dimX, idx+1});
                }
            }
        }

        size_t clothConstraintSize = sizeof(clothConstraints[0]) * clothConstraints.size();
        size_t clothBendingConstraintSize = sizeof(clothBendingConstraints[0]) * clothBendingConstraints.size();

        int distanceConstraintCount = clothConstraints.size();
        int bendingConstraintCount = clothBendingConstraints.size();

        hipMemcpy(d_constraintListCloth, clothConstraints.data(), clothConstraintSize, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintListClothBending, clothBendingConstraints.data(), clothBendingConstraintSize, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintCounterCloth, &distanceConstraintCount, sizeof(int) * 1, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintCounterClothBending, &bendingConstraintCount, sizeof(int) * 1, hipMemcpyHostToDevice);

        // box
        color = {1, 0, 0, 1};
        vec3 rot = {0,0,0};
        ivec3 dim = {10,10,10};
        vec3 pos = {-5, 0, -5};

        particleCountRB = dimX*dimZ;
        int objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color, false, 5);
        particleCountRB += dim.x() * dim.y() * dim.z();
    }

    if (scenario > 2 && scenario < 8)
        initRigidBodies(distance, scenario);

    if (scenario == 11) {
        vec3 rot = {0,0,0};
        ivec3 dim = {5,5,5};

        color = {1.0, 0., .0, 1};

        vec3 pos = {0, 10, 0};
        int objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
        particleCountRB += dim.x() * dim.y() * dim.z();
    }

    if (scenario > 2 && scenario != 6 && scenario != 7 && scenario < 7)
        deactivateNonRB<<<BLOCKS, BLOCK_SIZE>>>(d_particles);
    CUDA_SYNC_CHECK_ERROR();
    
    resetRigidBody<<<BLOCKS, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();

    caclulateRigidBodyOriginOfMass<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCountRB, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    initRigidBodyParticles<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCountRB, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();

    resetRigidBody<<<BLOCKS, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();
}

// 4.0
void ParticleSystem::initRigidBodies(float distance, int scenario) {
    // spawn
    ivec3 dim;
    vec3 pos;
    vec3 rot;
    vec4 color;
    int objParticleCount;

    if (scenario != 3 && scenario != 5 && scenario != 7) {
        color = {.8, .6, .5, 1};

        pos = linearRand(vec3(-40, 20, -40), vec3(40, 30, 40));
        rot = {0,0,0};
        objParticleCount = loadObj(rigidBodyCount++, particleCountRB, pos, rot, color);
        particleCountRB += objParticleCount;
        printf("%i\n", objParticleCount);

        pos = {0, 70, 0};
        rot = {0,0,0};
        objParticleCount = loadObj(rigidBodyCount++, particleCountRB, pos, rot, color);
        particleCountRB += objParticleCount;
    }

    color = {1.0, 0., .0, 1};

    if (scenario == 5) {
        rot = {0,0,0};
        dim = {5,5,5};

        pos = {0, 30, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
        particleCountRB += dim.x() * dim.y() * dim.z();

        pos = {0, 20, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color, false, 10);
        particleCountRB += dim.x() * dim.y() * dim.z();

        pos = {0, 10, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
        particleCountRB += dim.x() * dim.y() * dim.z();

    } else {

        for (int i = 0; i < 20; i++) {
            ivec3 dim = linearRand(ivec3(3,3,3), ivec3(5,5,5));
            vec3 pos = linearRand(vec3(-30, 10, -30), vec3(30, 40, 30));
            vec3 rot = linearRand(vec3(0, 0, 0), vec3(M_PI_2, M_PI_2, M_PI_2));
            //initCuboidParticles<<<1, 32>>>(d_particles, rigidBodyCount++, pos, dim, rot, color, particleCountRB, d_rigidBodies);
            //CUDA_SYNC_CHECK_ERROR();
            objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
            particleCountRB += dim.x() * dim.y() * dim.z();
        }

    }

}

// 1.2
// positive overlap
inline __device__ float collideSpherePlane(float r, vec3 pos, Saiga::Plane &plane) {
    return r - (pos.dot(plane.normal) - plane.d);
    //return plane.sphereOverlap(particle.position, particle.radius);
}

// 1.3
// positive overlap
inline __device__ float collideSphereSphere(float r1, float r2, vec3 pos1, vec3 pos2) {
    return (r1 + r2) - (pos1 - pos2).norm();
}

__global__ void resetConstraintCounter(int *constraintCounter, int *constraintCounterWalls) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1)
        return;
    *constraintCounter = 0;
    *constraintCounterWalls = 0;
}

__global__ void resetConstraints(int *constraints, int maxConstraintNum, int *constraintCounter, int *constraintCounterWalls) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id == 0) {
        *constraintCounter = 0;
        *constraintCounterWalls = 0;
    }
    if (ti.thread_id >= maxConstraintNum)
        return;
    
    constraints[ti.thread_id * 2 + 0] = -1;
    constraints[ti.thread_id * 2 + 1] = -1;
}

__global__ void resetCounter(int *counter) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1)
        return;
    *counter = 0;
}

__global__ void createConstraintParticles(Saiga::ArrayView<Particle>particles, int *constraints, int *constraintCounter, int maxConstraintNum) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle pa = particles[ti.thread_id];

    for (int i = ti.thread_id + 1; i < particles.size(); i++) {        
        Particle pb = particles[i];

        float d0 = collideSphereSphere(pa.radius, pb.radius, pa.predicted, pb.predicted);
        if (d0 > 0) {
            int idx = atomicAdd(constraintCounter, 1);
            if (idx >= maxConstraintNum - 1) {
                *constraintCounter = maxConstraintNum;
                return;
            }
            constraints[idx*2 + 0] = ti.thread_id;
            constraints[idx*2 + 1] = i;
        }
    }
}

__global__ void createConstraintWalls(Saiga::ArrayView<Particle> particles, Saiga::ArrayView<Saiga::Plane> walls, int *constraints, int *constraintCounter, int maxConstraintNum) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle p = particles[ti.thread_id];

    for (int i = 0; i < walls.size(); i++) {
        Saiga::Plane wall = walls[i];
        
        float d0 = collideSpherePlane(p.radius, p.predicted, wall);
        if (d0 > 0) {
            int idx = atomicAdd(constraintCounter, 1);
            if (idx >= maxConstraintNum - 1) {
                *constraintCounter = maxConstraintNum;
                return;
            }
            constraints[idx*2 + 0] = ti.thread_id;
            constraints[idx*2 + 1] = i;
        }
    }
}

__global__ void solverPBDParticlesSDF(Saiga::ArrayView<Particle> particles, int *constraints, int *constraintCounter, int maxConstraintNum, float relaxP, RigidBody *rigidBodies, float mu_k=0, float mu_s=0, float mu_f=0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxA = constraints[ti.thread_id*2 + 0];
    int idxB = constraints[ti.thread_id*2 + 1];
    Particle &pa = particles[idxA];
    Particle &pb = particles[idxB];

    if (pa.rbID == -2 && pb.rbID == -2) // deactivate for fluid
        return;

    ParticleCalc pa_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxA]), &pa_copy);
    ParticleCalc pb_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxB]), &pb_copy);

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = 1.0f; //pa.massinv;
    float m2 = 1.0f; //pb.massinv;

    float d = collideSphereSphere(pa_copy.radius, pb_copy.radius, pa_copy.predicted, pb_copy.predicted);
    vec3 n = (pa_copy.predicted - pb_copy.predicted).normalized();
    
    if (pa.rbID >= 0 || pb.rbID >= 0) {
        vec3 sdf1 = pa.sdf;
        vec3 sdf2 = pb.sdf;
        mat3 R;
        if (pa.rbID >= 0 && pb.rbID >= 0) {
            Particle pi;
            Particle pj;
            if (sdf1.norm() <= sdf2.norm()) {
                d = sdf1.norm();
                n = normalize(sdf1);
                R = rigidBodies[pa.rbID].A;
            } else {
                d = sdf2.norm();
                n = -normalize(sdf2);
                R = rigidBodies[pb.rbID].A;
            }
        } else if (pa.rbID >= 0) {
            d = sdf1.norm();
            n = normalize(sdf1);
            R = rigidBodies[pa.rbID].A;
        } else if (pb.rbID >= 0) {
            d = sdf2.norm();
            n = -normalize(sdf2);
            R = rigidBodies[pb.rbID].A;
        }
        n = R * -n;
        if (d <= 1.0) {
            // border particle
            d = collideSphereSphere(pa_copy.radius, pb_copy.radius, pa_copy.predicted, pb_copy.predicted);
            vec3 xij = -(pa_copy.predicted - pb_copy.predicted).normalized();
            if (xij.dot(n) < 0.f) {
                n = xij - 2.0f*(xij.dot(n))*n;
            } else {
                n = xij;
            }
        }
        n = -n;
    }

    float m = (m1 / (m1 + m2));
    vec3 dx1 = m * d * n;
    vec3 dx2 = - (1.0f - m) * d * n;

    // Friction
    if (mu_f) {
        //vec3 a = ((pa.predicted - pa.position) - (pb.predicted - pb.position));
        vec3 a = ((pa.position - pa.predicted) - (pb.position - pb.predicted));
        vec3 dx_orthogonal = a - (a.dot(n))*n; // a_orthogonal_n

        if (!dx_orthogonal.norm() < mu_s * d) {
            float min = mu_k * d / dx_orthogonal.norm();
            min = min <= 1.0 ? min : 1.0;
            dx_orthogonal *= min;
        }

        vec3 dx1_f = m * dx_orthogonal;
        vec3 dx2_f = - (1.0f - m) * dx_orthogonal;
        
        dx1 += dx1_f * mu_f;
        dx2 += dx2_f * mu_f;
    }
    // END Friction

    if (pa.fixed)
        dx2 *= 2.0;
    if (pb.fixed)
        dx1 *= 2.0;

    // jacobi integration
    if (!pa.fixed) {
        atomicAdd(&pa.d_predicted[0], dx1[0]);
        atomicAdd(&pa.d_predicted[1], dx1[1]);
        atomicAdd(&pa.d_predicted[2], dx1[2]);
    }
    if (!pb.fixed) {
        atomicAdd(&pb.d_predicted[0], dx2[0]);
        atomicAdd(&pb.d_predicted[1], dx2[1]);
        atomicAdd(&pb.d_predicted[2], dx2[2]);
    }
}

__global__ void solverPBDWalls(Saiga::ArrayView<Particle> particles, Saiga::ArrayView<Saiga::Plane> walls, int *constraints, int *constraintCounter, int maxConstraintNum, float relaxP, float mu_k=0, float mu_s=0, float mu_f=0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxP = constraints[ti.thread_id*2 + 0];
    int idxW = constraints[ti.thread_id*2 + 1];
    Particle &p = particles[idxP];
    Saiga::Plane &w = walls[idxW];

    if (p.fixed)
        return;

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = 1.0f; //p.massinv;
    float m2 = 0;
    float d = -collideSpherePlane(p.radius, p.predicted, w);
    //float d = -wall.sphereOverlap(particle.predicted, particle.radius);
    vec3 n = w.normal;
    float m = (m1 / (m1 + m2));
    vec3 dx1 = - m * d * n;

    // Friction
    if (mu_f) {
        //vec3 a = ((pa.predicted - pa.position) - (pb.predicted - pb.position));
        vec3 a = (p.position - p.predicted);
        vec3 dx_orthogonal = a - (a.dot(n))*n; // a_orthogonal_n

        if (!dx_orthogonal.norm() < mu_s * d) {
            float min = mu_k * d / dx_orthogonal.norm();
            min = min <= 1.0 ? min : 1.0;
            dx_orthogonal *= min;
        }

        vec3 dx1_f = m * dx_orthogonal;
        vec3 dx2_f = - (1.0f - m) * dx_orthogonal;
    
        dx1 += dx1_f * mu_f;
    }
    // END Friction

    atomicAdd(&p.d_predicted[0], dx1[0]);
    atomicAdd(&p.d_predicted[1], dx1[1]);
    atomicAdd(&p.d_predicted[2], dx1[2]);
}

// TODO Change to list sorted with by key
__global__ void updateLookupTable(Saiga::ArrayView<Particle> particles, int *particleIdLookup) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    particleIdLookup[particles[ti.thread_id].id] = ti.thread_id;
}

__global__ void solverPBDCloth(Saiga::ArrayView<Particle> particles, ClothConstraint *constraints, int *constraintCounter, int maxConstraintNum, int *particleIdLookup) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxA_ = constraints[ti.thread_id].first;
    int idxB_ = constraints[ti.thread_id].second;
    int idxA = particleIdLookup[idxA_];
    int idxB = particleIdLookup[idxB_];
    Particle &pa = particles[idxA];
    Particle &pb = particles[idxB];

    ParticleCalc pa_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxA]), &pa_copy);
    ParticleCalc pb_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxB]), &pb_copy);

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = 1.0f; //pa.massinv;
    float m2 = 1.0f; //pb.massinv;

    float d = collideSphereSphere(constraints[ti.thread_id].dist, 0, pa_copy.predicted, pb_copy.predicted);
    vec3 n = (pa_copy.predicted - pb_copy.predicted).normalized();
    float m = (m1 / (m1 + m2));
    vec3 dx1 = m * d * n;
    vec3 dx2 = - (1.0f - m) * d * n;

    if (pa.fixed)
        dx2 *= 2.0;
    if (pb.fixed)
        dx1 *= 2.0;

    // jacobi integration
    if (!pa.fixed) {
        atomicAdd(&pa.d_predicted[0], dx1[0]);
        atomicAdd(&pa.d_predicted[1], dx1[1]);
        atomicAdd(&pa.d_predicted[2], dx1[2]);
    }
    if (!pb.fixed) {
        atomicAdd(&pb.d_predicted[0], dx2[0]);
        atomicAdd(&pb.d_predicted[1], dx2[1]);
        atomicAdd(&pb.d_predicted[2], dx2[2]);
    }
}

__device__ void changePredicted(Particle &p, vec3 dx) {
    if (!p.fixed) {
        atomicAdd(&p.d_predicted[0], dx[0]);
        atomicAdd(&p.d_predicted[1], dx[1]);
        atomicAdd(&p.d_predicted[2], dx[2]);
    }
}

__global__ void solverPBDClothBending(Saiga::ArrayView<Particle> particles, ClothBendingConstraint *constraints, int *constraintCounter, int maxConstraintNum, int *particleIdLookup, float testFloat) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idx_[4] = { constraints[ti.thread_id].id1,
                    constraints[ti.thread_id].id2,
                    constraints[ti.thread_id].id3,
                    constraints[ti.thread_id].id4 };
    int idx[4] = {  particleIdLookup[idx_[0]],
                    particleIdLookup[idx_[1]],
                    particleIdLookup[idx_[2]],
                    particleIdLookup[idx_[3]] };

    vec3 p12 = (particles[idx[0]].predicted + particles[idx[1]].predicted) / 2.0f;
    //vec3 p12 = particles[idx[0]].predicted;

    vec3 p1 = particles[idx[0]].predicted - p12;
    vec3 p2 = particles[idx[1]].predicted - p12;
    vec3 p3 = particles[idx[2]].predicted - p12;
    vec3 p4 = particles[idx[3]].predicted - p12;

    vec3 n1 = (p2.cross(p3)).normalized();
    vec3 n2 = (p2.cross(p4)).normalized();

    float epsilon = 1e-5;

    if (n1.norm() < epsilon || n2.norm() < epsilon)
        return;

    float d = n1.dot(n2);
    d = d > 1.0f ? 1.0f : d;
    d = d < -1.0f ? -1.0f : d;


    vec3 q3 = (p2.cross(n2) + n1.cross(p2)*d) / (p2.cross(p3).norm());
    vec3 q4 = (p2.cross(n1) + n2.cross(p2)*d) / (p2.cross(p4).norm());
    vec3 q2 = - (p3.cross(n2) + n1.cross(p3)*d) / (p2.cross(p3).norm()) - (p4.cross(n1) + n2.cross(p4)*d) / (p2.cross(p4).norm());
    vec3 q1 = -q2-q3-q4;

    //if (q1.norm() < epsilon || q2.norm() < epsilon || q3.norm() < epsilon || q4.norm() < epsilon)
    //    return;

    float norm2_1 = q1.norm() * q1.norm();
    float norm2_2 = q2.norm() * q2.norm();
    float norm2_3 = q3.norm() * q3.norm();
    float norm2_4 = q4.norm() * q4.norm();

    const float omega1 = 1.0f;
    float angle0 = M_PI;
    float sqrt_d2 = sqrtf(1.0f-d*d);

    float sum_omega_q = norm2_1 + norm2_2 + norm2_3 + norm2_4;
    sum_omega_q *= omega1;

    if (sum_omega_q < epsilon)
        return;

    float dp = - (omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    dp *= testFloat;

    float dp1 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp2 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp3 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp4 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);

    changePredicted(particles[idx[0]], dp * q1);
    changePredicted(particles[idx[1]], dp * q2);
    changePredicted(particles[idx[2]], dp * q3);
    changePredicted(particles[idx[3]], dp * q4);
}

__global__ void reset_cell_list(std::pair<int, int>* cell_list, int cellCount) {
    Saiga::CUDA::ThreadInfo<> ti;

    if (ti.thread_id < cellCount) {
        cell_list[ti.thread_id].first = -1;
    }
}
// TODO remove one
__global__ void reset_cell_list_opti(std::pair<int, int>* cell_list, int cellCount, int particleCount) {
    Saiga::CUDA::ThreadInfo<> ti;

    if (ti.thread_id < cellCount) {
        cell_list[ti.thread_id].first = particleCount;
        cell_list[ti.thread_id].second = 0;
    }
}

__device__ ivec3 calculate_cell_idx(vec3 position, float cellSize) {
    return (position / cellSize).cast<int>(); // incorrect but faster
    /*vec3 idxf(position / cellSize);
    idxf = {floor(idxf[0]), floor(idxf[1]), floor(idxf[2])};
    return idxf.cast<int>();*/
}

/*__device__ int calculate_hash_idx(ivec3 cell_idx, ivec3 cell_dims, int cellCount) {
    int i2 = ((cell_idx.x() % cell_dims.x()) + cell_dims.x()) % cell_dims.x();
    int j2 = ((cell_idx.y() % cell_dims.y()) + cell_dims.y()) % cell_dims.y();
    int k2 = ((cell_idx.z() % cell_dims.z()) + cell_dims.z()) % cell_dims.z();
    int flat_cell_idx = i2 * cell_dims.y() * cell_dims.z() + j2 * cell_dims.z() + k2;
    return flat_cell_idx;
}*/
// TODO fails if position is too negative
__device__ int calculate_hash_idx(ivec3 cell_idx, ivec3 cell_dims, int cellCount) {
    int i2 = (cell_idx.x() + cell_dims.x()) % cell_dims.x();
    int j2 = (cell_idx.y() + cell_dims.y()) % cell_dims.y();
    int k2 = (cell_idx.z() + cell_dims.z()) % cell_dims.z();
    int flat_cell_idx = i2 * cell_dims.y() * cell_dims.z() + j2 * cell_dims.z() + k2;
    return flat_cell_idx;
}

__global__ void createLinkedCells(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ivec3 cell_idx = calculate_cell_idx(particles[ti.thread_id].position, cellSize);
        int flat_cell_idx = calculate_hash_idx(cell_idx, cell_dims, cellCount);
        particle_list[ti.thread_id] = atomicExch(&cell_list[flat_cell_idx].first, ti.thread_id);
    }
}

__global__ void calculateHash(Saiga::ArrayView<Particle> particles, int* particle_hash, std::pair<int, int>* cell_list, int* particle_list, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ivec3 cell_idx = calculate_cell_idx(particles[ti.thread_id].predicted, cellSize);
        int flat_cell_idx = calculate_hash_idx(cell_idx, cell_dims, cellCount);
        particle_hash[ti.thread_id] = flat_cell_idx;
    }
}

__global__ void createLinkedCellsOpti(Saiga::ArrayView<Particle> particles, int* particle_hash, std::pair<int, int>* cell_list, int* particle_list, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        int flat_cell_idx = particle_hash[ti.thread_id];
        //atomicMin(&cell_list[flat_cell_idx].first, ti.thread_id);
        //atomicAdd(&cell_list[flat_cell_idx].second, 1);

        // replace every
        //int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
        // with
        //int end_idx = cell_list[neighbor_flat_idx].second + 1;

        if (ti.thread_id > 0) {
            if (flat_cell_idx != particle_hash[ti.thread_id - 1])
                cell_list[flat_cell_idx].first = ti.thread_id;
        } else {
            cell_list[flat_cell_idx].first = ti.thread_id;
        }

        if (ti.thread_id < particles.size() - 1) {
            if (flat_cell_idx != particle_hash[ti.thread_id + 1])
                cell_list[flat_cell_idx].second = ti.thread_id;
        } else {
            cell_list[flat_cell_idx].second = ti.thread_id;
        }
    }
}

__global__ void createConstraintParticlesLinkedCellsRigidBodiesFluid(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;

    if (ti.thread_id < particles.size()) {
        //Particle pa = particles[ti.thread_id];
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);
        int rbIDa = particles[ti.thread_id].rbID;

        ivec3 cell_idx = calculate_cell_idx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here

        /*for (int x = -1; x <= 1; x++) {
        //for (int x = -1; x <= 0; x++) {
            for (int y = -1; y <= 1; y++) {
                //    if (x == 0 && y > 0)
                //        break;
                for (int z = -1; z <= 1; z++) {
                //    if (x == 0 && y == 0 && z > 0)
                //        break;
                }
            }
        }*/
        static const int X_CONSTS[14] = {-1,-1,-1,-1,-1,-1,-1,-1,-1, 0, 0, 0, 0, 0};
        static const int Y_CONSTS[14] = {-1,-1,-1, 0, 0, 0, 1, 1, 1,-1,-1,-1, 0, 0};
        static const int Z_CONSTS[14] = {-1, 0, 1,-1, 0, 1,-1, 0, 1,-1, 0, 1,-1, 0};

        for (int i = 0; i < 14; i++) {
            int x = X_CONSTS[i];
            int y = Y_CONSTS[i];
            int z = Z_CONSTS[i];
            
            ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
            int neighbor_flat_idx = calculate_hash_idx(neighbor_cell_idx, cell_dims, cellCount);
            int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
            int end_idx = cell_list[neighbor_flat_idx].second + 1;
            for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {

                int rbIDb = particles[neighbor_particle_idx].rbID;
                if ( (rbIDa == -1 || rbIDb == -1 || rbIDa != rbIDb) &&
                        (i != 13 || neighbor_particle_idx > ti.thread_id) ) {
                    //Particle pb = particles[neighbor_particle_idx];
                    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);
                    float d0 = collideSphereSphere(pa.radius, pb.radius, pa.predicted, pb.predicted);
                    if (d0 > 0) {
                        int idx = atomicAdd(constraintCounter, 1);
                        if (idx >= maxConstraintNum - 1) {
                            *constraintCounter = maxConstraintNum;
                            return;
                        }
                        constraints[idx*2 + 0] = ti.thread_id; // = tid
                        constraints[idx*2 + 1] = neighbor_particle_idx;
                    }
                }
                // Follow linked list
            }
        }
    }
}

// 6.2
__device__ __host__ float W_poly6(float r, float h) {
    if (r > h)
        return 0;
    float h2 = h * h;
    float hd = h2 - r * r;
    float hd3 = hd * hd * hd;
    float h3 = h2 * h;
    float h9 = h3 * h3 * h3;
    return 315.f / (64.f * M_PI * h9) * hd3;
}

__device__ __host__ vec3 W_spiky(vec3 r, float h, float epsilon) {
    float d = r.norm();
    if (d <= epsilon || d > h)
        return {0, 0, 0};
    float hd = h - d;
    //vec3 hd2 = r.normalized() * hd * hd;
    vec3 hd2 = r/d * hd * hd;
    float h3 = h * h * h;
    float h6 = h3 * h3;
    return -45.f / (M_PI * h6) * hd2;
}

inline __device__ __host__ float range(float value, float min, float max) {
    return value = value < min ? min : (value > max ? max : value);
}

__device__ float calculate_spray(float C_density, float rho0inv) {
    //float min_density = (1.0f * m) * rho0inv - 1.0;
    float min_density = 5 * rho0inv - 1.0; // 1 * W_poly(0, h) + 3 * W_poly(0.5, h)
    float max_density = 7.5 * rho0inv - 1.0; // 1 * W_poly(0, h) + 9 * W_poly(0.5, h) // 1.57 + x * 0.66; x= 3: 3.5, 6: 5.5, 9: 7.5
    //float spray = 1.0f - (C_density + 4.0f) / 2.0f;
    float non_spray = (C_density - min_density) / (max_density - min_density);
    non_spray = range(non_spray, 0, 1);
    float spray = 1.0f - (non_spray * non_spray);
    return spray;
}

__global__ void computeDensityAndLambda(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float omega_lambda_relax, float particleRadius) {
    Saiga::CUDA::ThreadInfo<> ti;

    const float m = 1.0;

    if (ti.thread_id < particles.size()) {
        //Particle pa = particles[ti.thread_id];
        int rbIDa = particles[ti.thread_id].rbID;
        if (rbIDa != -2)
            return;
        const float rho0inv = (8.0 * particleRadius * particleRadius * particleRadius);
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);

        ivec3 cell_idx = calculate_cell_idx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here

        float rho = 0;
        vec3 spiky_sum = {0, 0, 0};
        float lambda2 = 0;

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculate_hash_idx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + 1;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        //Particle pb = particles[neighbor_particle_idx];
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);
                        int rbIDb = particles[neighbor_particle_idx].rbID;
                        
                        vec3 d_p = pa.predicted - pb.predicted;

                        float d_rho = m * W_poly6((d_p).norm(), h);
                        rho += d_rho;

                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky) * rho0inv;
                        float spiky_norm = spiky.norm();
                        spiky_sum += spiky;
                        lambda2 += spiky_norm * spiky_norm;
                    }
                }
            }
        }

        float C_density = rho * rho0inv - 1.0;
        float lambda1 = spiky_sum.norm();
        lambda1 *= lambda1;
        float lambda = -C_density / (lambda1 + lambda2 + omega_lambda_relax);

        // gischt
        float spray = calculate_spray(C_density, rho0inv);
        vec4 water_color = {0, 0, 0.8, 1};
        vec4 spray_color = {1, 1, 1, 1};
        float old_spray = particles[ti.thread_id].color[0];
        float new_spray = spray;
        if (new_spray < old_spray)
            new_spray = old_spray * 0.995;
        particles[ti.thread_id].color = (1.0f - new_spray) * water_color + new_spray * spray_color;

        particles[ti.thread_id].lambda = lambda;
    }
}

__global__ void updateParticlesPBD2IteratorFluid(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float particleRadius, float artificial_pressure_k, int artificial_pressure_n, float w_poly_d_q) {
    Saiga::CUDA::ThreadInfo<> ti;


    if (ti.thread_id < particles.size()) {
        int rbIDa = particles[ti.thread_id].rbID;
        if (rbIDa != -2)
            return;
        const float rho0inv = (8.0 * particleRadius * particleRadius * particleRadius);
        //Particle pa = particles[ti.thread_id];
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);

        // 6 d
        float lambda1 = particles[ti.thread_id].lambda;

        ivec3 cell_idx = calculate_cell_idx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here

        vec3 lambda_spiky = {0, 0, 0};

        //float w_poly_d_q = W_poly6(delta_q * h, h);

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculate_hash_idx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + 1;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        int rbIDb = particles[neighbor_particle_idx].rbID;
                        if (rbIDb != -2)
                            continue;
                        //Particle pb = particles[neighbor_particle_idx];
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);

                        // 6 d
                        float lambda2 = particles[neighbor_particle_idx].lambda;
                                
                        vec3 d_p = pa.predicted - pb.predicted;
                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky);

                        // 6 e surface
                        float d_poly = W_poly6((d_p).norm(), h) / w_poly_d_q; // W_poly6(delta_q * h, h);
                        float poly = d_poly;
                        for (int i = 0; i < artificial_pressure_n - 1; i++) {
                            poly *= d_poly;
                        }
                        float s_corr = -artificial_pressure_k * poly;

                        // 6 d, e
                        vec3 d_lambda_spiky = (lambda1 + lambda2 + s_corr) * spiky;
                        lambda_spiky += d_lambda_spiky;
                    }
                }
            }
        }

        particles[ti.thread_id].d_predicted += lambda_spiky * rho0inv;
    }
}

__global__ void computeVorticityAndViscosity(float dt, Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float c_viscosity) {
    Saiga::CUDA::ThreadInfo<> ti;

    if (ti.thread_id < particles.size()) {
        //Particle& pa = particles[ti.thread_id];
        ParticleCalc1 pa;
        ParticleCalc1 pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc1*>(&particles[ti.thread_id].velocity), &pa);
        int rbIDa = pa.rbID;
        if (rbIDa != -2)
            return;


        ivec3 cell_idx = calculate_cell_idx(pa.position, cellSize); // actually pa.position but we only load predicted and its identical here

        vec3 curl = {0, 0, 0};

        vec3 viscosity = {0, 0, 0};

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculate_hash_idx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + 1;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        //Particle& pb = particles[neighbor_particle_idx];
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc1*>(&particles[neighbor_particle_idx].velocity), &pb);

                        int rbIDb = pb.rbID;
                        if (rbIDb != -2)
                            continue;

                        // 6 f 1
                        // vorticity
                        vec3 d_velocity = pb.velocity - pa.velocity;

                        vec3 d_p = pa.position - pb.position;
                                
                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky);
                        curl += d_velocity.cross(spiky);

                        // 6 g
                        // viscosity
                        float poly = W_poly6((d_p).norm(), h);
                        viscosity += d_velocity * poly;
                    }
                }
            }
        }

        particles[ti.thread_id].sdf = curl;

        particles[ti.thread_id].d_momentum = c_viscosity * viscosity;
    }
}

__device__ vec3 calculate_wind(vec3 pa, vec3 pb, vec3 wind_direction, float wind_speed) {
    float h = 1;
    vec3 UP = {0, 1, 0};
    vec3 d_p = pa - pb;
    if (d_p.norm() > h) // || d_p.x() * d_p.x() < 1e-5 || d_p.y() * d_p.y() < 1e-5)
        return {0,0,0};
    float wind_force = d_p.dot(wind_direction) * d_p.dot(UP); //d_p.x() * d_p.y();
    float wpoly = W_poly6((d_p).norm(), h) * wind_force;
    return UP * wind_force * wind_speed/10.0f;
}

__global__ void applyVorticityAndViscosity(float dt, Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float epsilon_vorticity, vec3 wind_direction, float wind_speed) {
    Saiga::CUDA::ThreadInfo<> ti;

    if (ti.thread_id < particles.size()) {
        //Particle& pa = particles[ti.thread_id];
        ParticleCalc2 pa;
        ParticleCalc3 pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc2*>(&particles[ti.thread_id].position), &pa);
        int rbIDa = pa.rbID;

        if (rbIDa != -2)
            return;


        ivec3 cell_idx = calculate_cell_idx(pa.position, cellSize); // actually pa.position but we only load predicted and its identical here

        vec3 curl_gradient = {0, 0, 0};

        vec3 d_velocity = {0, 0, 0};

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculate_hash_idx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + 1;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        //Particle& pb = particles[neighbor_particle_idx];
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc3*>(&particles[neighbor_particle_idx].position), &pb);
                        int rbIDb = pb.rbID;
                        if (rbIDb != -2)
                            continue;

                        if (neighbor_particle_idx == ti.thread_id)
                            continue;
                            
                        // 6 f
                        // vorticity
                        curl_gradient += pa.sdf.norm() * W_spiky(pa.position - pb.position, h, epsilon_spiky);

                        // wind and waves
                        d_velocity += calculate_wind(pa.position, pb.position, wind_direction, wind_speed);
                    }
                }
            }
        }

        vec3 force = epsilon_vorticity * curl_gradient.normalized().cross(pa.sdf);
        // apply vorticity force
        d_velocity += force * 1.0f; // pa.massinv; // TODO mass von material abhaengig machen nicht aus particle lesen
        // apply viscosity
        d_velocity += pa.d_momentum;


        // update velocity
        particles[ti.thread_id].velocity += d_velocity;

        // reset curl for sdf
        particles[ti.thread_id].sdf = {0,0,0};
    }
}

void ParticleSystem::update(float dt) {
    lastDt = dt;
    if (physicsMode == 0) {      
        resetConstraintCounter<<<1, 32>>>(d_constraintCounter, d_constraintCounterWalls);
        //resetConstraints<<<Saiga::CUDA::getBlockCount(maxConstraintNum, BLOCK_SIZE), BLOCK_SIZE>>>(d_constraintList, maxConstraintNum, d_constraintCounter, d_constraintCounterWalls);
        //CUDA_SYNC_CHECK_ERROR();

        const unsigned int BLOCKS_CELLS = Saiga::CUDA::getBlockCount(cellCount, BLOCK_SIZE);
        reset_cell_list_opti<<<BLOCKS_CELLS, BLOCK_SIZE>>>(d_cell_list, cellCount, particleCount);
        CUDA_SYNC_CHECK_ERROR();

        // moved up from previously after createConstraintWalls before iteration loop
        updateParticlesPBD1_radius<<<BLOCKS, BLOCK_SIZE>>>(dt, gravity, d_particles, dampV, particleRadiusWater, particleRadiusCloth);

        calculateHash<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particle_hash, d_cell_list, d_particle_list, cellDim, cellCount, cellSize);
        CUDA_SYNC_CHECK_ERROR();
        thrust::sort_by_key(thrust::device_pointer_cast(d_particle_hash), thrust::device_pointer_cast(d_particle_hash) + particleCount, d_particles.device_begin());

        createLinkedCellsOpti<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particle_hash, d_cell_list, d_particle_list, cellDim, cellCount, cellSize);
        //createLinkedCells<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, cellDim, cellCount, cellSize);
        CUDA_SYNC_CHECK_ERROR();

        createConstraintParticlesLinkedCellsRigidBodiesFluid<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize);

        createConstraintWalls<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_walls, d_constraintListWalls, d_constraintCounterWalls, maxConstraintNumWalls);
        CUDA_SYNC_CHECK_ERROR();

        float w_poly_d_q = W_poly6(delta_q * h, h);

        updateLookupTable<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particleIdLookup);

        float calculatedRelaxP = relaxP;
        for (int i = 0; i < solverIterations; i++) {
            // 6 b, c
            computeDensityAndLambda<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, omega_lambda_relax, particleRadiusRestDensity);
            CUDA_SYNC_CHECK_ERROR();
            // 6 d, e
            updateParticlesPBD2IteratorFluid<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, particleRadiusRestDensity, artificial_pressure_k, artificial_pressure_n, w_poly_d_q);
            CUDA_SYNC_CHECK_ERROR();

            if (useCalculatedRelaxP) {
                calculatedRelaxP = 1 - pow(1 - calculatedRelaxP, 1.0/(i+1));
            }
            // TODO N -> maxConstraintNum
            updateRigidBodies();
            solverPBDParticlesSDF<<<Saiga::CUDA::getBlockCount(maxConstraintNum, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintList, d_constraintCounter, maxConstraintNum, relaxP, d_rigidBodies, mu_k, mu_s, mu_f);

            solverPBDWalls<<<Saiga::CUDA::getBlockCount(maxConstraintNumWalls, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_walls, d_constraintListWalls, d_constraintCounterWalls, maxConstraintNumWalls, relaxP, mu_k, mu_s, mu_f);
            CUDA_SYNC_CHECK_ERROR();
            
            solverPBDCloth<<<Saiga::CUDA::getBlockCount(maxConstraintNumCloth, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintListCloth, d_constraintCounterCloth, maxConstraintNumCloth, d_particleIdLookup);
            if (testBool)
                solverPBDClothBending<<<Saiga::CUDA::getBlockCount(maxConstraintNumClothBending, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintListClothBending, d_constraintCounterClothBending, maxConstraintNumClothBending, d_particleIdLookup, testFloat);
            
            updateParticlesPBD2Iterator<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, calculatedRelaxP);
            CUDA_SYNC_CHECK_ERROR();
        }

        constraintsShapeMatchingRB();

        updateParticlesPBD2<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, relaxP);
        CUDA_SYNC_CHECK_ERROR();

        computeVorticityAndViscosity<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, c_viscosity);
        CUDA_SYNC_CHECK_ERROR();
        applyVorticityAndViscosity<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, epsilon_vorticity, wind_direction, wind_speed);
        CUDA_SYNC_CHECK_ERROR();
        
        hipDeviceSynchronize();
    }
    steps += 1;
}

// 2.3 Ray
__global__ void rayList(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    
    Particle &particle = particles[ti.thread_id];
    vec3 z = ray.origin - particle.position;
    float dz = ray.direction.dot(z);
    float Q = (dz * dz) - z.dot(z) + particle.radius * particle.radius;

    if (Q > 0) {
        int idx = atomicAdd(rayHitCount, 1);
        list[idx].first = ti.thread_id;
        list[idx].second = -dz;
    }
}

__global__ void rayColor(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, vec4 color) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    
    if (ti.thread_id == 0)
        particles[list[min].first].color = color;

    list[ti.thread_id].second = 0;
}

__global__ void rayImpulse(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    
    if (ti.thread_id == 0)
        particles[list[min].first].velocity += ray.direction * 42;

    list[ti.thread_id].second = 0;
}

__global__ void rayInflate(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, bool inflate, float maxParticleRadius) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    
    if (ti.thread_id == 0) {
        int idx = list[min].first;
        if (inflate) {
            if (particles[idx].radius * 2 > maxParticleRadius)
                return;
            particles[idx].radius *= 2;
            particles[idx].massinv /= 4;
        } else {
            particles[idx].radius /= 2;
            particles[idx].massinv *= 4;
        }
    }
    list[ti.thread_id].second = 0;
}

__global__ void rayRevert(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    
    if (ti.thread_id == 0) {
        int idx = list[min].first;
        particles[idx].radius = 0.5; // TODO radius
        particles[idx].velocity = {0,0,0};
        particles[idx].d_momentum = {0,0,0};
    }

    list[ti.thread_id].second = 0;
}

__global__ void rayExplosion(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, bool explode, float explodeMult) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    
    if (ti.thread_id == list[min].first)
        return;

    Particle &explodeParticle = particles[list[min].first];
    Particle &particle = particles[ti.thread_id];
    vec3 dir = particle.position - explodeParticle.position;
    float d = dir.norm();
    if (!explode)
        explodeMult = -explodeMult;
    if (d < 4) {
        particle.velocity += 1.0 / (d) * dir * explodeMult;
        particle.d_momentum += 1.0 / (d) * dir * explodeMult;
    }

    list[ti.thread_id].second = 0;
}

__global__ void raySplit(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, int splitCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;

    // TODO fix for fixed and different kinds of particles
    
    if (ti.thread_id == 0) {
        Particle &particle = particles[list[min].first];
        particle.radius /= 2;
        // 5 pseudo random indices
        float r = particle.radius;
        int randIdx = min * 1117 % 757 + (int)list[min].first % 137 + *rayHitCount % 17;

        for (int i = 0; i < splitCount; i++) {
            Particle &p = particles[(randIdx + i) % particles.size()];

            // reuse the current particle
            if (i == splitCount - 1) {
                p = particles[list[min].first];
            }

            // pseudo random offset
            int x = p.position[0] * 10.0 + p.velocity[0] * 100.0 + p.velocity[0] * 100.0;
            int y = p.position[1] * 10.0 + p.velocity[1] * 100.0 + p.velocity[1] * 100.0;
            int z = p.position[2] * 10.0 + p.velocity[2] * 100.0 + p.velocity[2] * 100.0;
            vec3 randOffset = vec3{x % 17, y % 17, z % 17} / 17;
            // min offset radius around original position
            if (randOffset[0] >= 0)
                randOffset[0] += r;
            else
                randOffset[0] -= r;
            if (randOffset[1] >= 0)
                randOffset[1] += r;
            else
                randOffset[1] -= r;
            if (randOffset[2] >= 0)
                randOffset[2] += r;
            else
                randOffset[2] -= r;
            // normalize
            randOffset /= 1 + r;
            // set attributes
            p.position = particle.position + randOffset * r * 2;
            p.predicted = particle.predicted + randOffset * r * 2;
            p.d_predicted = particle.d_predicted;
            p.radius = r;
            p.color = particle.color;
            p.velocity = particle.velocity;
        }
    }

    list[ti.thread_id].second = 0;
}

// remove if
struct remove_predicate
{
  __host__ __device__
  bool operator()(const thrust::pair<int, float> x)
  {
    return x.second <= 1e-5;
  }
};

// min element
struct compare_predicate
{
  __host__ __device__
  bool operator()(thrust::pair<int, float> a, thrust::pair<int, float> b)
  {
    return a.second < b.second;
  }
};

void ParticleSystem::ray(Saiga::Ray ray) {
    CUDA_SYNC_CHECK_ERROR();
    thrust::device_vector<thrust::pair<int, float>> d_vec(1000);
    //thrust::device_vector<float> d_vec2(1000);
    //resetCounter<<<1, 32>>>(d_rayHitCount);
    
    resetCounter<<<1, 32>>>(d_rayHitCount);
    CUDA_SYNC_CHECK_ERROR();
    rayList<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount);
    CUDA_SYNC_CHECK_ERROR();
    int N = thrust::remove_if(d_vec.begin(), d_vec.end(), remove_predicate()) - d_vec.begin();
    if (N == 0)
        return;
    int min = thrust::min_element(d_vec.begin(), d_vec.begin() + N, compare_predicate()) - d_vec.begin();

    if (actionMode == 0) {
        rayColor<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, color);
    } else if (actionMode == 1) {
        rayImpulse<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min);
    } else if (actionMode == 2) {
        rayExplosion<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, true, explosionForce);
    } else if (actionMode == 3) {
        rayExplosion<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, false, explosionForce);
    } else if (actionMode == 4) {
        raySplit<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, splitCount);
    } else if (actionMode == 5) {
        rayInflate<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, true, maxParticleRadius);
        /*CUDA_SYNC_CHECK_ERROR();
        update(lastDt);
        CUDA_SYNC_CHECK_ERROR();
        rayRevert<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min);
        CUDA_SYNC_CHECK_ERROR();*/
    } else if (actionMode == 6) {
        rayInflate<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, false, maxParticleRadius);
    }
    CUDA_SYNC_CHECK_ERROR();
}