#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
#include <cmath>

#include "particleSystem.h"
#include "saiga/core/util/assert.h"

//#include "saiga/cuda/cudaHelper.h"
#include "saiga/cuda/device_helper.h"

#include "saiga/cuda/memory.h"

// 4.0
#include <Eigen/Core>
#include "saiga/core/math/random.h"

#include "svd3_cuda.h"

// 4.4
#include "saiga/core/geometry/AccelerationStructure.h"
#include "saiga/core/geometry/intersection.h"

// time
float t = 0;

void ParticleSystem::setDevicePtr(void* particleVbo) {
    d_particles = ArrayView<Particle>((Particle*) particleVbo, particleCount);
}

__global__ void updateParticlesPBD1_radius(float dt, vec3 gravity, Saiga::ArrayView<Particle>particles, float damp_v, float particleRadiusWater, float particleRadiusCloth) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (p.fixed)
        return;

    /*
    // quite expensive memory access
    if (p.rbID == -2)
        p.radius = particleRadiusWater;
    else if (p.rbID == -3)
        p.radius = particleRadiusCloth;
    */

    vec3 newVelocity = p.velocity + dt * gravity;
    // dampVelocities
    newVelocity *= damp_v;

    p.predicted = p.position + dt * newVelocity;
}

__global__ void updateParticlesPBD2Iterator(float dt, Saiga::ArrayView<Particle>particles, float relax_p) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (!p.fixed) {
        p.predicted += relax_p * p.d_predicted;
    }
    // reset
    p.d_predicted = {0, 0, 0};
}
__global__ void updateParticlesPBD2(float dt, Saiga::ArrayView<Particle>particles, float relax_p) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    if (p.rbID == -4)
        return;

    if (!p.fixed) {
        //p.predicted += relax_p * p.d_predicted;
        p.velocity = (p.predicted - p.position) / dt;
        p.position = p.predicted;
    }
    // reset
    p.d_predicted = {0, 0, 0};

    // 6.2
    p.lambda = 0;
}

__global__ void resetOcean(Saiga::ArrayView<Particle> d_particles, int startId, int endId, int xMax, int zMax, vec3 corner, vec4 color, vec3 fluidDim) {
    Saiga::CUDA::ThreadInfo<> ti;
    int id = ti.thread_id;

    if (id >= startId && id < endId) {
        int y = (id - startId) / (xMax * zMax);
        int z = ((id - startId) - (y * xMax * zMax)) / xMax;
        int x = ((id - startId) - (y * xMax * zMax)) % xMax;

        Particle &p = d_particles[id];
        // for fluids
        int matId = -2;
        float distance = 0.5;
        float particleRenderRadius = 0.3;


        vec3 random_offset = vec3((id % 3) * 0.01, (id % 7) * 0.01, (id % 11) * 0.01);

        vec3 position = vec3(x, y, z) * distance + corner;
        // for trochoidals
        if ((position[0] < -fluidDim[0]/2) || (position[2] < -fluidDim[2]/2) || (position[0] > fluidDim[0]/2) || (position[2] > fluidDim[2]/2)) {
            matId = -4;
        }
        p.position = position + random_offset;
        p.velocity ={0, 0, 0};
        p.massinv = 1.0/1.0;
        p.predicted = p.position;
        // 2.3
        p.color = color;
        p.radius = particleRenderRadius;

        p.fixed = false;

        // 4.0
        p.rbID = matId;
        p.relative ={0, 0, 0};
        p.sdf ={0, 0, 0};


        // 6.0
        p.lambda = 0;

        p.id = ti.thread_id; // cloth

        if (matId == -4) {
            p.relative = p.position;
        }
    }

}

// a bit redundant
__global__ void resetParticlesStartEnd(Saiga::ArrayView<Particle> d_particles, Saiga::ArrayView<vec3> d_gradient, int startId, int endId, int xMax, int zMax, vec3 corner, float distance, int matId, vec4 color, float particleRenderRadius) {
    Saiga::CUDA::ThreadInfo<> ti;
    int id = ti.thread_id;

    if (id < d_particles.size() && id >= startId && id < endId) {
        Particle &p = d_particles[id];

        int y = (id - startId) / (xMax * zMax);
        int z = ((id - startId) - (y * xMax * zMax)) / xMax;
        int x = ((id - startId) - (y * xMax * zMax)) % xMax;
        float offset = d_particles[id].radius;

        vec3 random_offset = vec3((id % 3) * 0.01, (id % 7) * 0.01, (id % 11) * 0.01);

        p.position = vec3(x, y, z) * distance + corner + random_offset;

        p.velocity = {0, 0, 0};
        p.massinv = 1.0/1.0;
        p.predicted = p.position;
        // 2.3
        p.color = color;
        p.radius = particleRenderRadius;

        p.fixed = false;

        // 4.0
        p.rbID = matId;
        p.relative = {0, 0, 0};
        p.sdf = {0, 0, 0};
        if (matId == -4) {
            p.sdf = d_gradient[id - startId];
        } 

        // 6.0
        p.lambda = 0;

        p.id = ti.thread_id; // cloth

        if (matId == -4) {
            p.relative = p.position;
        }
    }

}

__global__ void resetParticles(int x, int z, vec3 corner, float distance, Saiga::ArrayView<Particle>particles, float randInitMul, float particleRenderRadius, int rbID, vec4 color) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &p = particles[ti.thread_id];

    int xPos = (ti.thread_id) % x;
    int zPos = ((ti.thread_id - xPos) / x) % z;
    int yPos = (((ti.thread_id - xPos) / x) - zPos) / z;
    vec3 pos = {xPos, yPos, zPos};

    // pseudo random position offset
    int rand = ti.thread_id + p.position[0];
    p.position = corner + pos * distance + vec3{rand % 11, rand % 17, rand % 13} * randInitMul;

    p.velocity = {0, 0, 0};
    p.massinv = 1.0/1.0;
    p.predicted = p.position;
    // 2.3
    p.color = color;
    p.radius = particleRenderRadius;

    p.fixed = false;

    // 4.0
    p.rbID = rbID;
    p.relative = {0,0,0};
    p.sdf = {0,0,0};

    // 6.0
    p.lambda = 0;

    p.id = ti.thread_id; // cloth

    if (rbID == -4) {
        p.relative = p.position;
    }
}

__global__ void initParticles(int startIdx, int count, int x, int z, vec3 corner, float distance, Saiga::ArrayView<Particle>particles, float randInitMul, float particleRenderRadius, int rbID, vec4 color, bool fixed=false, float mass=1.0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    if (ti.thread_id < startIdx || ti.thread_id >= startIdx + count)
        return;

    int idx = ti.thread_id - startIdx;
    Particle &p = particles[ti.thread_id];

    int xPos = (idx) % x;
    int zPos = ((idx - xPos) / x) % z;
    int yPos = (((idx - xPos) / x) - zPos) / z;
    vec3 pos = {xPos, yPos, zPos};

    // pseudo random position offset
    int rand = ti.thread_id + p.position[0];
    p.position = corner + pos * distance + vec3{rand % 11, rand % 17, rand % 13} * randInitMul;

    p.velocity = {0, 0, 0};
    p.massinv = 1.0f/mass;
    p.predicted = p.position;
    // 2.3
    p.color = color;
    p.radius = particleRenderRadius;

    p.fixed = fixed;

    // 4.0
    p.rbID = rbID;
    p.relative = {0,0,0};
    p.sdf = {0,0,0};

    // 6.0
    p.lambda = 0;
}

// 4.0
__global__ void initCuboidParticles(Saiga::ArrayView<Particle> particles, int id, vec3 pos, ivec3 dim, vec3 rot, vec4 color, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id > 0)
        return;
    
    mat3 rotMat;
    rotMat = Eigen::AngleAxisf(rot.x(), vec3::UnitX())
        * Eigen::AngleAxisf(rot.y(), vec3::UnitY())
        * Eigen::AngleAxisf(rot.z(), vec3::UnitZ());
    
    int count = dim.x() * dim.y() * dim.z();

    for (int i = 0; i < dim.x(); i++) {
        for (int j = 0; j < dim.y(); j++) {
            for (int k = 0; k < dim.z(); k++) {
                vec3 p = {i, j, k};
                p = rotMat * p;
                p += pos;
                particles[particleCountRB].position = p;
                particles[particleCountRB].predicted = p;
                particles[particleCountRB].rbID = id;

                particles[particleCountRB].color = color;

                //4.4
                ivec3 idx = {i, j, k};
                ivec3 dim2 = (dim/2);
                ivec3 dir;
                dir.x() = idx.x() < dim2.x() ? idx.x() - dim2.x() : dim2.x() - (dim.x() - idx.x() - 1);
                dir.y() = idx.y() < dim2.y() ? idx.y() - dim2.y() : dim2.y() - (dim.y() - idx.y() - 1);
                dir.z() = idx.z() < dim2.z() ? idx.z() - dim2.z() : dim2.z() - (dim.z() - idx.z() - 1);
                //float m = min(min(fabs(sdf.x()), fabs(sdf.y())), fabs(sdf.z()));

                ivec3 absdir = {abs(dir.x()), abs(dir.y()), abs(dir.z())};

                int minDir = max(max(absdir.x(), absdir.y()), absdir.z());
                vec3 sdf = {0,0,0};

                ivec3 dirSign = dir;
                dirSign.x() = dirSign.x() > 0 ? 1 : dirSign.x();
                dirSign.x() = dirSign.x() < 0 ? -1 : dirSign.x();
                dirSign.y() = dirSign.y() > 0 ? 1 : dirSign.y();
                dirSign.y() = dirSign.y() < 0 ? -1 : dirSign.y();
                dirSign.z() = dirSign.z() > 0 ? 1 : dirSign.z();
                dirSign.z() = dirSign.z() < 0 ? -1 : dirSign.z();

                if (absdir.x() == minDir)
                    sdf.x() = dirSign.x();
                if (absdir.y() == minDir)
                    sdf.y() = dirSign.y();
                if (absdir.z() == minDir)
                    sdf.z() = dirSign.z();

                int mx = min(i + 1, dim.x() - i);
                int my = min(j + 1, dim.y() - j);
                int mz = min(k + 1, dim.z() - k);

                float m = min(min(mx, my), mz);

                particles[particleCountRB].sdf = -m * normalize(sdf);// minus to point inwards

                printf("%i %i %i, %f, %f, %f, %f\n", i, j, k, sdf.x(), sdf.y(), sdf.z(), m);

                particleCountRB++;
            }
        }
    }

    rigidBodies[id].particleCount = count;
}

__global__ void initSingleRigidBodyParticle(Saiga::ArrayView<Particle> particles, int id, vec3 pos, vec3 sdf, vec4 color, int particleCountRB, RigidBody *rigidBodies, bool fixed=false, float mass=1.0, float particleRadius=0.5) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id > 0)
        return;
    
    particles[particleCountRB].position = pos;
    particles[particleCountRB].predicted = pos;
    particles[particleCountRB].rbID = id;

    particles[particleCountRB].color = color;

    particles[particleCountRB].fixed = fixed;
    particles[particleCountRB].massinv = 1.0f/mass;

    particles[particleCountRB].radius = particleRadius;

    // 4.4
    particles[particleCountRB].sdf = sdf;

    rigidBodies[id].particleCount++;
}

__global__ void initRigidBodyParticles(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    
    particles[ti.thread_id].relative = particles[ti.thread_id].predicted - rigidBodies[particles[ti.thread_id].rbID].originOfMass;
}

// 4.4
int ParticleSystem::loadObj(int rigidBodyCount, int particleCountRB, vec3 pos, vec3 rot, vec4 color) {
    Saiga::UnifiedModel model("objs/teapot.obj");
    Saiga::UnifiedMesh mesh = model.CombinedMesh().first;
    std::vector<Triangle> triangles = mesh.TriangleSoup();
    // 1
    Saiga::AABB bb = model.BoundingBox(); // mesh. or model.BoundingBox()
    vec3 min = bb.min;
    vec3 max = bb.max;
    // 2
    // Schnittstellen
    float maxObjParticleCount = 40;
    float maxSize = bb.maxSize();
    //float sampleDistance = 0.1;
    float sampleDistance = maxSize / maxObjParticleCount;
    int count = 0;
    Saiga::AccelerationStructure::ObjectMedianBVH omBVH(triangles);

    if (true) {
        // 3d voxel grid
        vec3 size = bb.Size() / sampleDistance;
        const int xDim = ceil(size.x());
        const int yDim = ceil(size.y());
        const int zDim = ceil(size.z());

        auto ***voxel = new std::pair<int, vec3>**[zDim];
        for(int i = 0; i < zDim; ++i) {
            voxel[i] = new std::pair<int, vec3>*[yDim];
            for(int j = 0; j < yDim; ++j) {
                voxel[i][j] = new std::pair<int, vec3>[xDim];
            }
        }
        // init voxels
        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    bool isInside = true;
                    for (float dx = -1; dx < 2; dx += 2) {
                        for (float dy = -1; dy < 2; dy += 2) {
                            for (float dz = -1; dz < 2; dz += 2) {
                                vec3 dir = {dx,dy,dz};
                                Saiga::Ray ray(dir, ori);
                                Saiga::Intersection::RayTriangleIntersection rti = omBVH.getClosest(ray);
                                if (!rti.valid)
                                    isInside = false;
                            }
                        }
                    }
                    if (isInside) {
                        count++;
                        voxel[z][y][x].first = 1;
                    } else {
                        voxel[z][y][x].first = 0;
                    }
                    // init border sdf
                    voxel[z][y][x].second = {0,0,0};
                    if (x == 0)
                        voxel[z][y][x].second[0] = +1;
                    else if (x == xDim-1)
                        voxel[z][y][x].second[0] = -1;
                    
                    if (y == 0)
                        voxel[z][y][x].second[1] = +1;
                    else if (y == yDim-1)
                        voxel[z][y][x].second[1] = -1;
                    
                    if (z == 0)
                        voxel[z][y][x].second[2] = +1;
                    else if (z == zDim-1)
                        voxel[z][y][x].second[2] = -1;
                }
            }
        }
        // calc distance field
        int i = 0;
        int changed = 1;
        while (changed) {
            i++;
            changed = 0;
            for (int x = 1; x < xDim-1; x++) {
                for (int y = 1; y < yDim-1; y++) {
                    for (int z = 1; z < zDim-1; z++) {
                        if (voxel[z][y][x].first == i) {
                            if (    voxel[z+1][y][x].first < i
                                ||  voxel[z-1][y][x].first < i
                                ||  voxel[z][y+1][x].first < i
                                ||  voxel[z][y-1][x].first < i
                                ||  voxel[z][y][x+1].first < i
                                ||  voxel[z][y][x-1].first < i)
                                continue;
                            voxel[z][y][x].first++;
                            changed++;
                        }
                    }
                }
            }
        }
        // calc derivative (normal)
        for (int x = 1; x < xDim-1; x++) {
            for (int y = 1; y < yDim-1; y++) {
                for (int z = 1; z < zDim-1; z++) {
                    if (voxel[z][y][x].first) {
                        float dz = voxel[z+1][y][x].first - voxel[z-1][y][x].first;
                        float dy = voxel[z][y+1][x].first - voxel[z][y-1][x].first;
                        float dx = voxel[z][y][x+1].first - voxel[z][y][x-1].first;
                        voxel[z][y][x].second = {dx, dy, dz};
                    }
                }
            }
        }

        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    if (voxel[z][y][x].first) {
                        count++;
                        float scaling = 1.0f;
                        vec3 position = pos + ori*(scaling / sampleDistance);
                        vec3 sdf = (float)voxel[z][y][x].first * normalize(voxel[z][y][x].second);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, sdf, color, particleCountRB++, d_rigidBodies);
                    }
                }
            }
        }

        for(int i = 0; i < zDim; ++i) {
            for(int j = 0; j < yDim; ++j) {
                delete [] voxel[i][j];
            }
            delete [] voxel[i];
        }
        delete [] voxel;

    } else {
        
        for (float x = min.x(); x < max.x(); x += sampleDistance) {
            for (float y = min.y(); y < max.y(); y += sampleDistance) {
                for (float z = min.z(); z < max.z(); z += sampleDistance) {
                    vec3 ori = {x,y,z};
                    bool isInside = true;
                    for (float dx = -1; dx < 2; dx += 2) {
                        for (float dy = -1; dy < 2; dy += 2) {
                            for (float dz = -1; dz < 2; dz += 2) {
                                vec3 dir = {dx,dy,dz};
                                Saiga::Ray ray(dir, ori);
                                Saiga::Intersection::RayTriangleIntersection rti = omBVH.getClosest(ray);
                                if (!rti.valid)
                                    isInside = false;
                            }
                        }
                    }
                    if (isInside) {
                        count++;
                        float scaling = 1.0f;
                        vec3 position = pos + ori * (scaling / sampleDistance);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, vec3{0.f,0.f,0.f}, color, particleCountRB++, d_rigidBodies);
                    }
                }
            }
        }

    }
    return count;
}

// 4.4
int ParticleSystem::loadBox(int rigidBodyCount, int particleCountRB, ivec3 dim, vec3 pos, vec3 rot, vec4 color, bool fixed=false, float mass=1.0, float scaling=1.0, float particleRadius=0.5) {    
    vec3 min = {0,0,0};
    int count = 0;
    float sampleDistance = 1.0;

        // 3d voxel grid
        //vec3 size = bb.Size() / sampleDistance;
        const int xDim = dim.x();
        const int yDim = dim.y();
        const int zDim = dim.z();

        auto ***voxel = new std::pair<int, vec3>**[zDim];
        for(int i = 0; i < zDim; ++i) {
            voxel[i] = new std::pair<int, vec3>*[yDim];
            for(int j = 0; j < yDim; ++j) {
                voxel[i][j] = new std::pair<int, vec3>[xDim];
            }
        }
        // init voxels
        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    
                    voxel[z][y][x].first = 1;

                    // init border sdf
                    voxel[z][y][x].second = {0,0,0};
                    if (x == 0)
                        voxel[z][y][x].second[0] = +1;
                    else if (x == xDim-1)
                        voxel[z][y][x].second[0] = -1;
                    
                    if (y == 0)
                        voxel[z][y][x].second[1] = +1;
                    else if (y == yDim-1)
                        voxel[z][y][x].second[1] = -1;
                    
                    if (z == 0)
                        voxel[z][y][x].second[2] = +1;
                    else if (z == zDim-1)
                        voxel[z][y][x].second[2] = -1;
                }
            }
        }
        // calc distance field
        int i = 0;
        int changed = 1;
        while (changed) {
            i++;
            changed = 0;
            for (int x = 1; x < xDim-1; x++) {
                for (int y = 1; y < yDim-1; y++) {
                    for (int z = 1; z < zDim-1; z++) {
                        if (voxel[z][y][x].first == i) {
                            if (    voxel[z+1][y][x].first < i
                                ||  voxel[z-1][y][x].first < i
                                ||  voxel[z][y+1][x].first < i
                                ||  voxel[z][y-1][x].first < i
                                ||  voxel[z][y][x+1].first < i
                                ||  voxel[z][y][x-1].first < i)
                                continue;
                            voxel[z][y][x].first++;
                            changed++;
                        }
                    }
                }
            }
        }
        // calc derivative (normal)
        for (int x = 1; x < xDim-1; x++) {
            for (int y = 1; y < yDim-1; y++) {
                for (int z = 1; z < zDim-1; z++) {
                    if (voxel[z][y][x].first) {
                        float dz = voxel[z+1][y][x].first - voxel[z-1][y][x].first;
                        float dy = voxel[z][y+1][x].first - voxel[z][y-1][x].first;
                        float dx = voxel[z][y][x+1].first - voxel[z][y][x-1].first;
                        voxel[z][y][x].second = {dx, dy, dz};
                    }
                }
            }
        }

        for (int x = 0; x < xDim; x++) {
            for (int y = 0; y < yDim; y++) {
                for (int z = 0; z < zDim; z++) {
                    vec3 ori = min + sampleDistance * ivec3{x, y, z}.cast<float>();
                    if (voxel[z][y][x].first) {
                        count++;
                        //float scaling = 0.5f;
                        vec3 position = pos + ori*(scaling / sampleDistance);
                        vec3 sdf = (float)voxel[z][y][x].first * normalize(voxel[z][y][x].second);
                        initSingleRigidBodyParticle<<<1, 32>>>(d_particles, rigidBodyCount, position, sdf, color, particleCountRB++, d_rigidBodies, fixed, mass, particleRadius);
                    }
                }
            }
        }

        for(int i = 0; i < zDim; ++i) {
            for(int j = 0; j < yDim; ++j) {
                delete [] voxel[i][j];
            }
            delete [] voxel[i];
        }
        delete [] voxel;

    return count;
}

__global__ void caclulateRigidBodyOriginOfMass(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        vec3 d_originOfMass = p.predicted / (float)rigidBodies[p.rbID].particleCount;
        atomicAdd(&rigidBodies[p.rbID].originOfMass[0], d_originOfMass[0]);
        atomicAdd(&rigidBodies[p.rbID].originOfMass[1], d_originOfMass[1]);
        atomicAdd(&rigidBodies[p.rbID].originOfMass[2], d_originOfMass[2]);
    }
}

__global__ void covariance(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        //vec3 pc = p.position - rigidBodies[p.rbID].originOfMass;
        mat3 pcr = (p.predicted - rigidBodies[p.rbID].originOfMass) * p.relative.transpose();

        atomicAdd(&rigidBodies[p.rbID].A(0,0), pcr(0,0));
        atomicAdd(&rigidBodies[p.rbID].A(0,1), pcr(0,1));
        atomicAdd(&rigidBodies[p.rbID].A(0,2), pcr(0,2));
        atomicAdd(&rigidBodies[p.rbID].A(1,0), pcr(1,0));
        atomicAdd(&rigidBodies[p.rbID].A(1,1), pcr(1,1));
        atomicAdd(&rigidBodies[p.rbID].A(1,2), pcr(1,2));
        atomicAdd(&rigidBodies[p.rbID].A(2,0), pcr(2,0));
        atomicAdd(&rigidBodies[p.rbID].A(2,1), pcr(2,1));
        atomicAdd(&rigidBodies[p.rbID].A(2,2), pcr(2,2));
    }
}

__global__ void SVD(RigidBody *rigidBodies, int rigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= rigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    rb.A = svd3_cuda::pd(rb.A);
}

__global__ void resolveRigidBodyConstraints(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particleCountRB)
        return;
    Particle &p = particles[ti.thread_id];
    if (p.rbID >= 0) {
        // dx = (Q*r + c) - p
        p.predicted += (rigidBodies[p.rbID].A * p.relative + rigidBodies[p.rbID].originOfMass) - p.predicted;
    }
}

__global__ void resetRigidBody(RigidBody *rigidBodies, int rigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= rigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    // reset
    rb.originOfMass = {0,0,0};
    rb.A = mat3::Zero().cast<float>();
}

__global__ void resetRigidBodyComplete(RigidBody *rigidBodies, int maxRigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= maxRigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    // reset
    rb.particleCount = 0;
    rb.originOfMass = {0,0,0};
    rb.A = mat3::Zero().cast<float>();
}

__global__ void initRigidBodiesRotation(RigidBody *rigidBodies, int rigidBodyCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= rigidBodyCount)
        return;
    RigidBody &rb = rigidBodies[ti.thread_id];
    rb.initA = rb.A;
}

void ParticleSystem::constraintsShapeMatchingRB() {
    updateRigidBodies();

    resolveRigidBodyConstraints<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();    
}

void ParticleSystem::updateRigidBodies() {
    const unsigned int BLOCKS_RB = Saiga::CUDA::getBlockCount(rigidBodyCount, BLOCK_SIZE);

    resetRigidBody<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();

    caclulateRigidBodyOriginOfMass<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    covariance<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    SVD<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();
    
}

// sehr haesslich
__global__ void deactivateNonRB(Saiga::ArrayView<Particle> particles) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    
    Particle &p = particles[ti.thread_id];
    if (p.rbID < 0) {
        p.position[1] += 1000000.0f;
        p.predicted[1] = p.position[1];
    }
}

std::vector<vec3> computeGradient(vec3 voxelGridEnd, int*** grid) {
    std::vector<vec3> gradients;
    std::vector<int> magnitudes;

    // This order must be consistent with the spawn order
    // (Otherwise output gradient in a 3d array)
    for (int y = 0; y < voxelGridEnd[1]; y+= 1) {
        for (int z = 0; z < voxelGridEnd[2]; z+= 1) {
            for (int x = 0; x < voxelGridEnd[0]; x += 1) {

                if (grid[x][y][z] == 0) {
                    continue;
                }
                vec3 grad = vec3(0, 0, 0);

                int left = 0;
                if (x > 0) {
                    left = grid[x-1][y][z];
                }
                int top = 0;
                if (y > 0) {
                    top = grid[x][y-1][z];
                }
                int back = 0;
                if (z > 0) {
                    back = grid[x][y][z-1];
                }

                int right = 0;
                if (x < voxelGridEnd[0]-1) {
                    right = grid[x+1][y][z];
                }
                int bottom = 0;
                if (y < voxelGridEnd[1]-1) {
                    bottom = grid[x][y+1][z];
                }
                int front = 0;
                if (z < voxelGridEnd[2]-1) {
                    front = grid[x][y][z+1];
                }

                // using central differencing
                grad[0] += (left-right);
                grad[1] += (top-bottom);
                grad[2] += (back-front);

                grad = normalize(grad);
                if (grad == vec3(0, 0, 0)) {
                    grad = vec3(0, 1, 0);
                }
                grad *= grid[x][y][z];

                // gradient output as a normal
                gradients.push_back(grad * (-1));
            }
        }
    }


    return gradients;
}

void computeSDF(vec3 voxelGridEnd, int*** grid) {

    // dynamic programming algorithm

    // go forward
    for (int x = 0; x < voxelGridEnd[0]; x += 1) {
        for (int y = 0; y < voxelGridEnd[1]; y+= 1) {
            for (int z = 0; z < voxelGridEnd[2]; z+= 1) {
                if (grid[x][y][z] == 0) {
                    continue;
                }
                if (x == 0 || y == 0 || z == 0) {
                    continue;
                }

                // look at -1 neighbors (already updated)
                int minNeighbor = std::min({grid[x-1][y][z], grid[x][y-1][z], grid[x][y][z-1]});
                grid[x][y][z] = minNeighbor + 1;
            }
        }
    }

    // go backward and overwrite wrong values in-place
    for (int x = voxelGridEnd[0]-1; x >= 0; x -= 1) {
        for (int y = voxelGridEnd[1]-1; y >= 0; y -= 1) {
            for (int z = voxelGridEnd[2]-1; z >= 0; z -= 1) {
                if (grid[x][y][z] == 0) {
                    continue;
                }
                if (x == voxelGridEnd[0]-1 || y == voxelGridEnd[1]-1 || z == voxelGridEnd[2]-1) {
                    grid[x][y][z] = 1;
                    continue;
                }

                // look at +1 neighbors (already updated)
                int minNeighbor =  std::min({grid[x+1][y][z], grid[x][y+1][z], grid[x][y][z+1]});
                grid[x][y][z] = min(grid[x][y][z], minNeighbor + 1);
            }
        }
    }
}

void ParticleSystem::reset(int x, int z, vec3 corner, float distance, float randInitMul, int scenario, vec3 fluidDim, vec3 trochoidal1Dim, vec3 trochoidal2Dim, ivec2 layers) {
    int rbID = -1; // free particles
    vec4 color = {0.0f, 1.0f, 0.0f, 1.f};
    if (scenario >= 7) {
        color = {0.1f, 0.2f, 0.8f, 1.f};
        rbID = -2; // fluid
    }
    if (scenario == 12) { // trochoidal test scenario
        color ={0.1f, 0.2f, 0.8f, 1.f};
        rbID = -4; // trochoidal particles
    }

    if (scenario == 13) {
        // scene parameters
        wave_number = 5;
        steepness = 0.2;
        wind_speed = 0;

        ivec3 trochDim = ivec3(20, 8, 20);
        int startId = 0;
        int endId = trochDim[0] * trochDim[1] * trochDim[2];
        vec3 voxelGridEnd = vec3(trochDim[0], trochDim[1], trochDim[2]);
        int ***grid3D = new int**[endId];
        for (int i = 0; i < trochDim[0]; i++) {
            grid3D[i] = new int*[trochDim[1]];
            for (int j = 0; j < trochDim[1]; j++) {
                grid3D[i][j] = new int[trochDim[2]];
                for (int h = 0; h < trochDim[2]; h++) {
                    grid3D[i][j][h] = 1;
                }
            }
        }

        // compute sdf for trochoidal ocean "block"
        computeSDF(voxelGridEnd, grid3D);
        std::vector<vec3> gradients = computeGradient(voxelGridEnd, grid3D);

        vec3* gradPtr;
        hipMalloc((void **)&gradPtr, sizeof(vec3) * trochDim[0] * trochDim[1] * trochDim[2]);
        hipMemcpy(gradPtr, gradients.data(), sizeof(vec3) * trochDim[0] * trochDim[1] * trochDim[2], hipMemcpyHostToDevice);
        ArrayView<vec3> d_gradient = make_ArrayView(gradPtr, trochDim[0] * trochDim[1] * trochDim[2]);

        // adds trochoidal particles
        resetParticlesStartEnd<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_gradient, startId, endId, x, z, vec3(-10, 0, -4), 0.4, -4, color, 0.3);
        CUDA_SYNC_CHECK_ERROR();

        startId = endId;
        endId = 20 * 20 * 40 * 2;

        // adds fluid particles
        resetParticlesStartEnd<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_gradient, startId, endId, x, z, corner + vec3(x, 0, 0), distance, -2, color, 0.5);
        CUDA_SYNC_CHECK_ERROR();
    }
    else if (scenario == 14) {
        // scene parameters
        wave_number = 3.5;
        steepness = 0.6;
        wind_direction = {-1.0, 0.0, -1.0};
        wind_speed = 0.7;
        solver_iterations = 1;
        c_viscosity = 0.02;
        epsilon_vorticity = 0.001;
        float distance = 0.5;

        // adds trochoidal particles
        // generate first layers of fluids and trochoidals
        resetOcean<<<BLOCKS, BLOCK_SIZE>>>(d_particles, 0, layers[0], x, z, corner, color, fluidDim);
        CUDA_SYNC_CHECK_ERROR();

        // generate top layer of fluids and trochoidals
        corner -= vec3(trochoidal2Dim[0], 0, trochoidal2Dim[2]);
        float height = trochoidal1Dim[1] - trochoidal2Dim[1];
        corner += vec3(0, height, 0);
        x += 1/distance * trochoidal2Dim[0] * 2;
        z += 1/distance * trochoidal2Dim[2] * 2;
        resetOcean<<<BLOCKS, BLOCK_SIZE>>>(d_particles, layers[0], layers[1], x, z, corner, color, fluidDim);
        CUDA_SYNC_CHECK_ERROR();
    }
    else {
        resetParticles<<<BLOCKS, BLOCK_SIZE>>>(x, z, corner, distance, d_particles, randInitMul, particleRenderRadius, rbID, color);
        CUDA_SYNC_CHECK_ERROR();
    }


    if (scenario == 9) {
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(0, 20*20*20, 20, 20, {-20, 0, -20}, distance, d_particles, randInitMul, particleRenderRadius, -2, {0.f, 0.f, 1.f, .1f}, false);
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(20*20*20, 40*40*40, 40, 40, {0, 0, 0}, distance, d_particles, randInitMul, particleRenderRadius, -2, {0.f, .2f, .7f, .3f}, false);
        CUDA_SYNC_CHECK_ERROR();
    }
    
    const unsigned int BLOCKS_RB = Saiga::CUDA::getBlockCount(maxRigidBodyCount, BLOCK_SIZE);
    resetRigidBodyComplete<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, maxRigidBodyCount);

    particleCountRB = 0;
    rigidBodyCount = 0;

    if (scenario == 7) {
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000, 10*5*10, 10, 5, {-2, 0, -2}, 1, d_particles, randInitMul, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
        initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000+10*5*10, 64, 4, 4, {0, 0, 10}, 1, d_particles, randInitMul, particleRenderRadius, -1, {.0f, .9f, .0f, 1.f}, false);
        CUDA_SYNC_CHECK_ERROR();
    }

    if (scenario == 8) {
        //initParticles<<<BLOCKS, BLOCK_SIZE>>>(10000, 19*2*10, 2, 19, {-1, 0, -20}, 1, d_particles, 0, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
        //initParticles<<<BLOCKS, BLOCK_SIZE>>>(11000, 19*2*10, 2, 19, {-1, 0, 1.5}, 1, d_particles, 0, particleRenderRadius, -1, {.0f, .0f, .0f, 1.f}, true);
    }

    if (scenario == 10) { // cloth
        rbID = -3; // free particles
        vec4 color = {1.0f, 1.0f, 1.0f, 1.f};
        resetParticles<<<BLOCKS, BLOCK_SIZE>>>(x, z, corner, distance, d_particles, randInitMul, particleRenderRadius, rbID, color);
        CUDA_SYNC_CHECK_ERROR();

        std::vector<ClothConstraint> clothConstraints(0);

        std::vector<ClothBendingConstraint> clothBendingConstraints(0);

        int dimX = 50;
        int dimZ = 50;

        for (int j = 0; j < dimZ; j++) {
            for (int i = 0; i < dimX; i++) {
                int idx = j * dimX + i;
                if (i < dimX - 1) {
                    clothConstraints.push_back({idx, idx+1, 1.0f * distance});
                }
                if (j < dimZ - 1) {
                    clothConstraints.push_back({idx, idx+dimX, 1.0f * distance});
                }
                if (j < dimZ - 1 && i < dimX - 1) {
                    if (i+j % 2)
                        clothConstraints.push_back({idx, idx+dimX+1, 1.4142f*distance});
                    else
                        clothConstraints.push_back({idx+dimX, idx+1, 1.4142f*distance});

                    clothBendingConstraints.push_back({idx+dimX+1, idx, idx+dimX, idx+1});
                }
            }
        }

        size_t clothConstraintSize = sizeof(clothConstraints[0]) * clothConstraints.size();
        size_t clothBendingConstraintSize = sizeof(clothBendingConstraints[0]) * clothBendingConstraints.size();

        int distanceConstraintCount = clothConstraints.size();
        int bendingConstraintCount = clothBendingConstraints.size();

        hipMemcpy(d_constraintListCloth, clothConstraints.data(), clothConstraintSize, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintListClothBending, clothBendingConstraints.data(), clothBendingConstraintSize, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintCounterCloth, &distanceConstraintCount, sizeof(int) * 1, hipMemcpyHostToDevice);
        hipMemcpy(d_constraintCounterClothBending, &bendingConstraintCount, sizeof(int) * 1, hipMemcpyHostToDevice);

        // box
        color = {1, 0, 0, 1};
        vec3 rot = {0,0,0};
        ivec3 dim = {10,10,10};
        vec3 pos = {-5, 0, -5};

        particleCountRB = dimX*dimZ;
        int objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color, false, 5);
        particleCountRB += dim.x() * dim.y() * dim.z();
    }

    if (scenario > 2 && scenario < 8)
        initRigidBodies(distance, scenario);

    if (scenario == 11 || scenario == 14) {
        vec3 rot = {0,0,0};
        ivec3 dim = {5,5,5};

        color = {1.0, 0., .0, 1};

        vec3 pos = {0, 10, 0};
        int objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color, false, 0.2, 0.5, 0.3);
        particleCountRB += dim.x() * dim.y() * dim.z();
    }

    if (scenario > 2 && scenario != 6 && scenario != 7 && scenario < 7)
        deactivateNonRB<<<BLOCKS, BLOCK_SIZE>>>(d_particles);
    CUDA_SYNC_CHECK_ERROR();
    
    resetRigidBody<<<BLOCKS, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();

    caclulateRigidBodyOriginOfMass<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCountRB, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();
    initRigidBodyParticles<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCountRB, d_rigidBodies);
    CUDA_SYNC_CHECK_ERROR();

    updateRigidBodies();
    initRigidBodiesRotation<<<BLOCKS_RB, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);

    resetRigidBody<<<BLOCKS, BLOCK_SIZE>>>(d_rigidBodies, rigidBodyCount);
    CUDA_SYNC_CHECK_ERROR();
}

// 4.0
void ParticleSystem::initRigidBodies(float distance, int scenario) {
    // spawn
    ivec3 dim;
    vec3 pos;
    vec3 rot;
    vec4 color;
    int objParticleCount;

    if (scenario != 3 && scenario != 5 && scenario != 7) {
        color = {.8, .6, .5, 1};

        pos = linearRand(vec3(-40, 20, -40), vec3(40, 30, 40));
        rot = {0,0,0};
        objParticleCount = loadObj(rigidBodyCount++, particleCountRB, pos, rot, color);
        particleCountRB += objParticleCount;
        printf("%i\n", objParticleCount);

        pos = {0, 70, 0};
        rot = {0,0,0};
        objParticleCount = loadObj(rigidBodyCount++, particleCountRB, pos, rot, color);
        particleCountRB += objParticleCount;
    }

    color = {1.0, 0., .0, 1};

    if (scenario == 5) {
        rot = {0,0,0};
        dim = {5,5,5};

        pos = {0, 30, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
        particleCountRB += dim.x() * dim.y() * dim.z();

        pos = {0, 20, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color, false, 10);
        particleCountRB += dim.x() * dim.y() * dim.z();

        pos = {0, 10, 0};
        objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
        particleCountRB += dim.x() * dim.y() * dim.z();

    } else {

        for (int i = 0; i < 20; i++) {
            ivec3 dim = linearRand(ivec3(3,3,3), ivec3(5,5,5));
            vec3 pos = linearRand(vec3(-30, 10, -30), vec3(30, 40, 30));
            vec3 rot = linearRand(vec3(0, 0, 0), vec3(M_PI_2, M_PI_2, M_PI_2));
            //initCuboidParticles<<<1, 32>>>(d_particles, rigidBodyCount++, pos, dim, rot, color, particleCountRB, d_rigidBodies);
            //CUDA_SYNC_CHECK_ERROR();
            objParticleCount = loadBox(rigidBodyCount++, particleCountRB, dim, pos, rot, color);
            particleCountRB += dim.x() * dim.y() * dim.z();
        }

    }

}

// 1.2
// positive overlap
inline __device__ float collideSpherePlane(float r, vec3 pos, Saiga::Plane &plane) {
    return r - (pos.dot(plane.normal) - plane.d);
    //return plane.sphereOverlap(particle.position, particle.radius);
}

// 1.3
// positive overlap
inline __device__ float collideSphereSphere(float r1, float r2, vec3 pos1, vec3 pos2) {
    return (r1 + r2) - (pos1 - pos2).norm();
}

__global__ void resetConstraintCounter(int *constraintCounter, int *constraintCounterWalls) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1)
        return;
    *constraintCounter = 0;
    *constraintCounterWalls = 0;
}

__global__ void resetConstraints(int *constraints, int maxConstraintNum, int *constraintCounter, int *constraintCounterWalls) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id == 0) {
        *constraintCounter = 0;
        *constraintCounterWalls = 0;
    }
    if (ti.thread_id >= maxConstraintNum)
        return;
    
    constraints[ti.thread_id * 2 + 0] = -1;
    constraints[ti.thread_id * 2 + 1] = -1;
}

__global__ void resetCounter(int *counter) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1)
        return;
    *counter = 0;
}

__global__ void createConstraintWalls(Saiga::ArrayView<Particle> particles, Saiga::ArrayView<Saiga::Plane> walls, int *constraints, int *constraintCounter, int maxConstraintNum) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle p = particles[ti.thread_id];

    for (int i = 0; i < walls.size(); i++) {
        Saiga::Plane wall = walls[i];
        
        float d0 = collideSpherePlane(p.radius, p.predicted, wall);
        if (d0 > 0) {
            int idx = atomicAdd(constraintCounter, 1);
            if (idx >= maxConstraintNum - 1) {
                *constraintCounter = maxConstraintNum;
                return;
            }
            constraints[idx*2 + 0] = ti.thread_id;
            constraints[idx*2 + 1] = i;
        }
    }
}

__global__ void solverPBDParticlesSDF(Saiga::ArrayView<Particle> particles, int *constraints, int *constraintCounter, int maxConstraintNum, float relax_p, RigidBody *rigidBodies, float mu_k=0, float mu_s=0, float mu_f=0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxA = constraints[ti.thread_id*2 + 0];
    int idxB = constraints[ti.thread_id*2 + 1];
    Particle &pa = particles[idxA];
    Particle &pb = particles[idxB];

    if (pa.rbID == -2 && pb.rbID == -2) // deactivate for fluid
        return;

    if (pa.rbID == -4 && pb.rbID == -4) // deactivate for trochoidal particles
        return;

    ParticleCalc pa_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxA]), &pa_copy);
    ParticleCalc pb_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxB]), &pb_copy);

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = pa.massinv;
    float m2 = pb.massinv;

    float d = collideSphereSphere(pa_copy.radius, pb_copy.radius, pa_copy.predicted, pb_copy.predicted);
    vec3 n = (pa_copy.predicted - pb_copy.predicted).normalized();
    
    if (pa.rbID >= 0 || pb.rbID >= 0) {
        vec3 sdf1 = pa.sdf;
        vec3 sdf2 = pb.sdf;
        mat3 R;
        if (pa.rbID >= 0 && pb.rbID >= 0) {
            Particle pi;
            Particle pj;
            if (sdf1.norm() <= sdf2.norm()) {
                d = sdf1.norm();
                n = normalize(sdf1);
                R = rigidBodies[pa.rbID].A;
            } else {
                d = sdf2.norm();
                n = -normalize(sdf2);
                R = rigidBodies[pb.rbID].A;
            }
        } else if (pa.rbID >= 0) {
            d = sdf1.norm();
            n = normalize(sdf1);
            R = rigidBodies[pa.rbID].A;
        } else if (pb.rbID >= 0) {
            d = sdf2.norm();
            n = -normalize(sdf2);
            R = rigidBodies[pb.rbID].A;
        }
        n = R * -n;
        if (d <= 1.0) {
            // border particle
            d = collideSphereSphere(pa_copy.radius, pb_copy.radius, pa_copy.predicted, pb_copy.predicted);
            vec3 xij = -(pa_copy.predicted - pb_copy.predicted).normalized();
            if (xij.dot(n) < 0.f) {
                n = xij - 2.0f*(xij.dot(n))*n;
            } else {
                n = xij;
            }
        }
        n = -n;
    }

    float m = (m1 / (m1 + m2));
    vec3 dx1 = m * d * n;
    vec3 dx2 = - (1.0f - m) * d * n;

    // Friction
    if (mu_f) {
        vec3 a = ((pa.position - pa.predicted) - (pb.position - pb.predicted)); //vec3 a = ((pa.predicted - pa.position) - (pb.predicted - pb.position));
        vec3 dx_orthogonal = a - (a.dot(n))*n; // a_orthogonal_n

        if (!dx_orthogonal.norm() < mu_s * d) {
            float min = mu_k * d / dx_orthogonal.norm();
            min = min <= 1.0 ? min : 1.0;
            dx_orthogonal *= min;
        }

        vec3 dx1_f = m * dx_orthogonal;
        vec3 dx2_f = - (1.0f - m) * dx_orthogonal;
        
        dx1 += dx1_f * mu_f;
        dx2 += dx2_f * mu_f;
    }
    // END Friction

    if (pa.fixed)
        dx2 *= 2.0;
    if (pb.fixed)
        dx1 *= 2.0;

    // jacobi integration
    if (!pa.fixed) {
        atomicAdd(&pa.d_predicted[0], dx1[0]);
        atomicAdd(&pa.d_predicted[1], dx1[1]);
        atomicAdd(&pa.d_predicted[2], dx1[2]);
    }
    if (!pb.fixed) {
        atomicAdd(&pb.d_predicted[0], dx2[0]);
        atomicAdd(&pb.d_predicted[1], dx2[1]);
        atomicAdd(&pb.d_predicted[2], dx2[2]);
    }
}

__global__ void solverPBDWalls(Saiga::ArrayView<Particle> particles, Saiga::ArrayView<Saiga::Plane> walls, int *constraints, int *constraintCounter, int maxConstraintNum, float relax_p, float mu_k=0, float mu_s=0, float mu_f=0) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxP = constraints[ti.thread_id*2 + 0];
    int idxW = constraints[ti.thread_id*2 + 1];
    Particle &p = particles[idxP];
    Saiga::Plane &w = walls[idxW];

    if (p.fixed)
        return;

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = p.massinv;
    float m2 = 0;
    float d = -collideSpherePlane(p.radius, p.predicted, w);
    //float d = -wall.sphereOverlap(particle.predicted, particle.radius);
    vec3 n = w.normal;
    float m = (m1 / (m1 + m2));
    vec3 dx1 = - m * d * n;

    // Friction
    if (mu_f) {
        vec3 a = (p.position - p.predicted); //vec3 a = ((pa.predicted - pa.position) - (pb.predicted - pb.position));
        vec3 dx_orthogonal = a - (a.dot(n))*n; // a_orthogonal_n

        if (!dx_orthogonal.norm() < mu_s * d) {
            float min = mu_k * d / dx_orthogonal.norm();
            min = min <= 1.0 ? min : 1.0;
            dx_orthogonal *= min;
        }

        vec3 dx1_f = m * dx_orthogonal;
        vec3 dx2_f = - (1.0f - m) * dx_orthogonal;
    
        dx1 += dx1_f * mu_f;
    }
    // END Friction

    atomicAdd(&p.d_predicted[0], dx1[0]);
    atomicAdd(&p.d_predicted[1], dx1[1]);
    atomicAdd(&p.d_predicted[2], dx1[2]);
}

__global__ void updateLookupTable(Saiga::ArrayView<Particle> particles, int *particleIdLookup) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    particleIdLookup[particles[ti.thread_id].id] = ti.thread_id;
}

__global__ void solverPBDCloth(Saiga::ArrayView<Particle> particles, ClothConstraint *constraints, int *constraintCounter, int maxConstraintNum, int *particleIdLookup) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idxA_ = constraints[ti.thread_id].first;
    int idxB_ = constraints[ti.thread_id].second;
    int idxA = particleIdLookup[idxA_];
    int idxB = particleIdLookup[idxB_];
    Particle &pa = particles[idxA];
    Particle &pb = particles[idxB];

    ParticleCalc pa_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxA]), &pa_copy);
    ParticleCalc pb_copy;
    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[idxB]), &pb_copy);

    // TODO mass von material abhaengig machen nicht aus particle lesen
    float m1 = pa.massinv;
    float m2 = pb.massinv;

    float d = collideSphereSphere(constraints[ti.thread_id].dist, 0, pa_copy.predicted, pb_copy.predicted);
    vec3 n = (pa_copy.predicted - pb_copy.predicted).normalized();
    float m = (m1 / (m1 + m2));
    vec3 dx1 = m * d * n;
    vec3 dx2 = - (1.0f - m) * d * n;

    if (pa.fixed)
        dx2 *= 2.0;
    if (pb.fixed)
        dx1 *= 2.0;

    // jacobi integration
    if (!pa.fixed) {
        atomicAdd(&pa.d_predicted[0], dx1[0]);
        atomicAdd(&pa.d_predicted[1], dx1[1]);
        atomicAdd(&pa.d_predicted[2], dx1[2]);
    }
    if (!pb.fixed) {
        atomicAdd(&pb.d_predicted[0], dx2[0]);
        atomicAdd(&pb.d_predicted[1], dx2[1]);
        atomicAdd(&pb.d_predicted[2], dx2[2]);
    }
}

__device__ void changePredicted(Particle &p, vec3 dx) {
    if (!p.fixed) {
        atomicAdd(&p.d_predicted[0], dx[0]);
        atomicAdd(&p.d_predicted[1], dx[1]);
        atomicAdd(&p.d_predicted[2], dx[2]);
    }
}

__global__ void solverPBDClothBending(Saiga::ArrayView<Particle> particles, ClothBendingConstraint *constraints, int *constraintCounter, int maxConstraintNum, int *particleIdLookup, float test_float) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= *constraintCounter || ti.thread_id >= maxConstraintNum)
        return;
    int idx_[4] = { constraints[ti.thread_id].id1,
                    constraints[ti.thread_id].id2,
                    constraints[ti.thread_id].id3,
                    constraints[ti.thread_id].id4 };
    int idx[4] = {  particleIdLookup[idx_[0]],
                    particleIdLookup[idx_[1]],
                    particleIdLookup[idx_[2]],
                    particleIdLookup[idx_[3]] };

    vec3 p12 = (particles[idx[0]].predicted + particles[idx[1]].predicted) / 2.0f;
    //vec3 p12 = particles[idx[0]].predicted;

    vec3 p1 = particles[idx[0]].predicted - p12;
    vec3 p2 = particles[idx[1]].predicted - p12;
    vec3 p3 = particles[idx[2]].predicted - p12;
    vec3 p4 = particles[idx[3]].predicted - p12;

    vec3 n1 = (p2.cross(p3)).normalized();
    vec3 n2 = (p2.cross(p4)).normalized();

    float epsilon = 1e-5;

    if (n1.norm() < epsilon || n2.norm() < epsilon)
        return;

    float d = n1.dot(n2);
    d = d > 1.0f ? 1.0f : d;
    d = d < -1.0f ? -1.0f : d;

    vec3 q3 = (p2.cross(n2) + n1.cross(p2)*d) / (p2.cross(p3).norm());
    vec3 q4 = (p2.cross(n1) + n2.cross(p2)*d) / (p2.cross(p4).norm());
    vec3 q2 = - (p3.cross(n2) + n1.cross(p3)*d) / (p2.cross(p3).norm()) - (p4.cross(n1) + n2.cross(p4)*d) / (p2.cross(p4).norm());
    vec3 q1 = -q2-q3-q4;

    //if (q1.norm() < epsilon || q2.norm() < epsilon || q3.norm() < epsilon || q4.norm() < epsilon)
    //    return;

    float norm2_1 = q1.norm() * q1.norm();
    float norm2_2 = q2.norm() * q2.norm();
    float norm2_3 = q3.norm() * q3.norm();
    float norm2_4 = q4.norm() * q4.norm();

    const float omega1 = 1.0f;
    float angle0 = M_PI;
    float sqrt_d2 = sqrtf(1.0f-d*d);

    float sum_omega_q = norm2_1 + norm2_2 + norm2_3 + norm2_4;
    sum_omega_q *= omega1;

    if (sum_omega_q < epsilon)
        return;

    float dp = - (omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    dp *= test_float;

    float dp1 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp2 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp3 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);
    float dp4 = -(omega1 * sqrt_d2 * (acosf(d) - angle0)) / (sum_omega_q);

    changePredicted(particles[idx[0]], dp * q1);
    changePredicted(particles[idx[1]], dp * q2);
    changePredicted(particles[idx[2]], dp * q3);
    changePredicted(particles[idx[3]], dp * q4);
}

__global__ void resetCellListOptimized(std::pair<int, int>* cell_list, int cellCount, int particleCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < cellCount) {
        cell_list[ti.thread_id].first = particleCount;
        cell_list[ti.thread_id].second = 0;
    }
}

__device__ ivec3 calculateCellIdx(vec3 position, float cellSize) {
    return (position / cellSize).cast<int>(); // incorrect but faster
    /*vec3 idxf(position / cellSize);
    idxf = {floor(idxf[0]), floor(idxf[1]), floor(idxf[2])};
    return idxf.cast<int>();*/
}

__device__ int calculateHashIdx(ivec3 cell_idx, ivec3 cell_dims, int cellCount) {
    int i2 = ((cell_idx.x() % cell_dims.x()) + cell_dims.x()) % cell_dims.x();
    int j2 = ((cell_idx.y() % cell_dims.y()) + cell_dims.y()) % cell_dims.y();
    int k2 = ((cell_idx.z() % cell_dims.z()) + cell_dims.z()) % cell_dims.z();
    int flat_cell_idx = i2 * cell_dims.y() * cell_dims.z() + j2 * cell_dims.z() + k2;
    return flat_cell_idx;
}

__global__ void calculateHash(Saiga::ArrayView<Particle> particles, int* particle_hash, std::pair<int, int>* cell_list, int* particle_list, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ivec3 cell_idx = calculateCellIdx(particles[ti.thread_id].predicted, cellSize);
        int flat_cell_idx = calculateHashIdx(cell_idx, cell_dims, cellCount);
        particle_hash[ti.thread_id] = flat_cell_idx;
    }
}

__global__ void createLinkedCellsOptimized(Saiga::ArrayView<Particle> particles, int* particle_hash, std::pair<int, int>* cell_list, int* particle_list, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        int flat_cell_idx = particle_hash[ti.thread_id];
        atomicMin(&cell_list[flat_cell_idx].first, ti.thread_id);
        atomicAdd(&cell_list[flat_cell_idx].second, 1);
    }
}

__global__ void createConstraintParticlesLinkedCellsRigidBodiesFluid(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);
        int rbIDa = particles[ti.thread_id].rbID;

        if (rbIDa == -4)
            return;

        ivec3 cell_idx = calculateCellIdx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here

        static const int X_CONSTS[14] = {-1,-1,-1,-1,-1,-1,-1,-1,-1, 0, 0, 0, 0, 0};
        static const int Y_CONSTS[14] = {-1,-1,-1, 0, 0, 0, 1, 1, 1,-1,-1,-1, 0, 0};
        static const int Z_CONSTS[14] = {-1, 0, 1,-1, 0, 1,-1, 0, 1,-1, 0, 1,-1, 0};

        for (int i = 0; i < 14; i++) {
            int x = X_CONSTS[i];
            int y = Y_CONSTS[i];
            int z = Z_CONSTS[i];
            
            ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
            int neighbor_flat_idx = calculateHashIdx(neighbor_cell_idx, cell_dims, cellCount);
            int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
            int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
            for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {

                int rbIDb = particles[neighbor_particle_idx].rbID;
                if (rbIDb == -4)
                    continue;
                if ( (rbIDa == -1 || rbIDb == -1 || rbIDa != rbIDb) &&
                        (i != 13 || neighbor_particle_idx > ti.thread_id) ) {
                    Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);
                    float d0 = collideSphereSphere(pa.radius, pb.radius, pa.predicted, pb.predicted);
                    if (d0 > 0) {
                        int idx = atomicAdd(constraintCounter, 1);
                        if (idx >= maxConstraintNum - 1) {
                            *constraintCounter = maxConstraintNum;
                            return;
                        }
                        constraints[idx*2 + 0] = ti.thread_id; // = tid
                        constraints[idx*2 + 1] = neighbor_particle_idx;
                    }
                }
                // Follow linked list
            }
        }
    }
}

// 6.2
__device__ __host__ float W_poly6(float r, float h) {
    if (r > h)
        return 0;
    float h2 = h * h;
    float hd = h2 - r * r;
    float hd3 = hd * hd * hd;
    float h3 = h2 * h;
    float h9 = h3 * h3 * h3;
    return 315.f / (64.f * M_PI * h9) * hd3;
}

__device__ __host__ vec3 W_spiky(vec3 r, float h, float epsilon) {
    float d = r.norm();
    if (d <= epsilon || d > h)
        return {0, 0, 0};
    float hd = h - d;
    vec3 hd2 = r/d * hd * hd; //vec3 hd2 = r.normalized() * hd * hd;
    float h3 = h * h * h;
    float h6 = h3 * h3;
    return -45.f / (M_PI * h6) * hd2;
}

inline __device__ __host__ float range(float value, float min, float max) {
    return value = value < min ? min : (value > max ? max : value);
}

__device__ float calculateSpray(float C_density, float rho0inv) {
    //float min_density = (1.0f * m) * rho0inv - 1.0;
    float min_density = 5 * rho0inv - 1.0; // 1 * W_poly(0, h) + 3 * W_poly(0.5, h)
    float max_density = 7.5 * rho0inv - 1.0; // 1 * W_poly(0, h) + 9 * W_poly(0.5, h) // 1.57 + x * 0.66; x= 3: 3.5, 6: 5.5, 9: 7.5
    float non_spray = (C_density - min_density) / (max_density - min_density);
    non_spray = range(non_spray, 0, 1);
    float spray = 1.0f - (non_spray * non_spray);
    return spray;
}

__global__ void computeDensityAndLambda(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float omega_lambda_relax, float particleRadius) {
    Saiga::CUDA::ThreadInfo<> ti;
    const float m = 1.0;

    if (ti.thread_id < particles.size()) {
        //Particle pa = particles[ti.thread_id];
        int rbIDa = particles[ti.thread_id].rbID;
        if (rbIDa != -2)
            return;
        const float rho0inv = (8.0 * particleRadius * particleRadius * particleRadius);
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);

        ivec3 cell_idx = calculateCellIdx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here
        float rho = 0;
        vec3 spiky_sum = {0, 0, 0};
        float lambda2 = 0;

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculateHashIdx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);
                        //int rbIDb = particles[neighbor_particle_idx].rbID;
                        
                        vec3 d_p = pa.predicted - pb.predicted;

                        float d_rho = m * W_poly6((d_p).norm(), h);
                        rho += d_rho;

                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky) * rho0inv;
                        float spiky_norm = spiky.norm();
                        spiky_sum += spiky;
                        lambda2 += spiky_norm * spiky_norm;
                    }
                }
            }
        }
        // compute density and lambda
        float C_density = rho * rho0inv - 1.0;
        float lambda1 = spiky_sum.norm();
        lambda1 *= lambda1;
        float lambda = -C_density / (lambda1 + lambda2 + omega_lambda_relax);
        particles[ti.thread_id].lambda = lambda;

        // gischt (spray)
        float spray = calculateSpray(C_density, rho0inv);
        vec4 water_color = {0, 0, 0.8, 1};
        vec4 spray_color = {1, 1, 1, 1};
        float old_spray = particles[ti.thread_id].color[0];
        float new_spray = spray;
        if (new_spray < old_spray)
            new_spray = old_spray * 0.995;
        particles[ti.thread_id].color = (1.0f - new_spray) * water_color + new_spray * spray_color;
    }
}

__global__ void updateParticlesPBD2IteratorFluid(Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float particleRadius, float artificial_pressure_k, int artificial_pressure_n, float w_poly_d_q) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        int rbIDa = particles[ti.thread_id].rbID;
        if (rbIDa != -2)
            return;
        ParticleCalc pa;
        ParticleCalc pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[ti.thread_id]), &pa);

        const float rho0inv = (8.0 * particleRadius * particleRadius * particleRadius);
        float lambda1 = particles[ti.thread_id].lambda;
        vec3 lambda_spiky = {0, 0, 0};
        //float w_poly_d_q = W_poly6(delta_q * h, h);
        ivec3 cell_idx = calculateCellIdx(pa.predicted, cellSize); // actually pa.position but we only load predicted and its identical here

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculateHashIdx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        int rbIDb = particles[neighbor_particle_idx].rbID;
                        if (!(rbIDb == -2 || rbIDb == -4))
                            continue;
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc*>(&particles[neighbor_particle_idx]), &pb);

                        // 6 d
                        float lambda2 = particles[neighbor_particle_idx].lambda;
                                
                        vec3 d_p = pa.predicted - pb.predicted;
                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky);

                        // 6 e surface
                        float d_poly = W_poly6((d_p).norm(), h) / w_poly_d_q; // W_poly6(delta_q * h, h);
                        float poly = d_poly;
                        for (int i = 0; i < artificial_pressure_n - 1; i++) {
                            poly *= d_poly;
                        }
                        float s_corr = -artificial_pressure_k * poly;

                        // 6 d, e
                        vec3 d_lambda_spiky = (lambda1 + lambda2 + s_corr) * spiky;
                        lambda_spiky += d_lambda_spiky;
                    }
                }
            }
        }
        particles[ti.thread_id].d_predicted += lambda_spiky * rho0inv;
    }
}

__global__ void computeVorticityAndViscosity(float dt, Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float c_viscosity) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ParticleCalc1 pa;
        ParticleCalc1 pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc1*>(&particles[ti.thread_id].velocity), &pa);
        int rbIDa = pa.rbID;
        if (rbIDa != -2)
            return;

        ivec3 cell_idx = calculateCellIdx(pa.position, cellSize); // actually pa.position but we only load predicted and its identical here
        vec3 curl = {0, 0, 0};
        vec3 viscosity = {0, 0, 0};

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculateHashIdx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc1*>(&particles[neighbor_particle_idx].velocity), &pb);
                        int rbIDb = pb.rbID;
                        if (!(rbIDb == -2 || rbIDb == -4))
                            continue;

                        // vorticity
                        vec3 d_velocity = pb.velocity - pa.velocity;
                        vec3 d_p = pa.position - pb.position;
                        vec3 spiky = W_spiky(d_p, h, epsilon_spiky);
                        curl += d_velocity.cross(spiky);
                        // viscosity
                        float poly = W_poly6((d_p).norm(), h);
                        viscosity += d_velocity * poly;
                    }
                }
            }
        }
        // compute verticity
        particles[ti.thread_id].sdf = curl;
        // compute velocity change by viscosity
        particles[ti.thread_id].d_momentum = c_viscosity * viscosity;
    }
}

__device__ vec3 calculateWind(vec3 pa, vec3 pb, vec3 wind_direction, float wind_speed) {
    float h = 1;
    vec3 UP = {0, 1, 0};
    vec3 d_p = pa - pb;
    if (d_p.norm() > h) // || d_p.x() * d_p.x() < 1e-5 || d_p.y() * d_p.y() < 1e-5)
        return {0,0,0};
    float wind_force = d_p.dot(wind_direction) * d_p.dot(UP); //d_p.x() * d_p.y();
    float wpoly = W_poly6((d_p).norm(), h) * wind_force;
    return UP * wind_force * wind_speed/10.0f;
}

__global__ void applyVorticityAndViscosity(float dt, Saiga::ArrayView<Particle> particles, std::pair<int, int>* cell_list, int* particle_list, int *constraints, int *constraintCounter, int maxConstraintNum, ivec3 cell_dims, int cellCount, float cellSize, float h, float epsilon_spiky, float epsilon_vorticity, vec3 wind_direction, float wind_speed) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < particles.size()) {
        ParticleCalc2 pa;
        ParticleCalc3 pb;
        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc2*>(&particles[ti.thread_id].position), &pa);
        int rbIDa = pa.rbID;
        if (rbIDa != -2)
            return;

        ivec3 cell_idx = calculateCellIdx(pa.position, cellSize); // actually pa.position but we only load predicted and its identical here
        vec3 curl_gradient = {0, 0, 0};
        vec3 d_velocity = {0, 0, 0};

        for (int x = -1; x <= 1; x++) {
            for (int y = -1; y <= 1; y++) {
                for (int z = -1; z <= 1; z++) {
                    ivec3 neighbor_cell_idx = cell_idx + ivec3(x, y, z);
                    int neighbor_flat_idx = calculateHashIdx(neighbor_cell_idx, cell_dims, cellCount);
                    int neighbor_particle_idx = cell_list[neighbor_flat_idx].first;
                    int end_idx = cell_list[neighbor_flat_idx].second + neighbor_particle_idx;
                    for (; neighbor_particle_idx < end_idx; neighbor_particle_idx++) {
                        Saiga::CUDA::vectorCopy(reinterpret_cast<ParticleCalc3*>(&particles[neighbor_particle_idx].position), &pb);
                        int rbIDb = pb.rbID;
                        if (!(rbIDb == -2 || rbIDb == -4))
                            continue;
                        if (neighbor_particle_idx == ti.thread_id)
                            continue;

                        // vorticity
                        curl_gradient += pa.sdf.norm() * W_spiky(pa.position - pb.position, h, epsilon_spiky);
                        // wind and waves
                        d_velocity += calculateWind(pa.position, pb.position, wind_direction, wind_speed);
                    }
                }
            }
        }
        vec3 force = epsilon_vorticity * curl_gradient.normalized().cross(pa.sdf);
        // apply vorticity force
        d_velocity += force * pa.massinv; // TODO mass von material abhaengig machen nicht aus particle lesen
        // apply viscosity
        d_velocity += pa.d_momentum;
        // update velocity
        particles[ti.thread_id].velocity += d_velocity;
        // reset curl for sdf
        particles[ti.thread_id].sdf = {0,0,0};
    }
}

__device__ vec3 trochoidalWaveOffset(vec3 gridPoint, vec2 direction, float wave_length, float steepness, float t) {
    direction = normalize(direction);
    float x = gridPoint[0];
    float y = gridPoint[1];
    float z = gridPoint[2];

    float k = 2 * M_PI / wave_length;
    // compute speed of waves
    float c = 9.8 / (k * 2.5);

    // amplitude
    float a = (steepness / 10 * y) / k;

    float f = k * (direction[0] * x + direction[1] * z - c * t);

    //float sin_f = sinf(f);
    //float cos_f = cosf(f);
    float sin_f;
    float cos_f;
    sincosf(f, &sin_f, &cos_f);

    float xOffset = direction[0] * a * sin_f;
    float yOffset = -a * cos_f;
    float zOffset = direction[1] * a * sin_f;

    return vec3(xOffset, yOffset, zOffset);
}

__global__ void updateTrochoidalParticles(Saiga::ArrayView<Particle> d_particles, float wave_length, float phase_speed, float steepness, float t) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id < d_particles.size()) {
        if (d_particles[ti.thread_id].rbID != -4) {
            return;
        }

        vec3 position = d_particles[ti.thread_id].relative;

        // add different trochoidal waves
        // main wave
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(1, 0), wave_length, steepness, t);
        // small waves
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.2, 0.8), wave_length * 0.8, steepness * 0.8, t);
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.8, 0.2), wave_length * 0.5, steepness * 0.95, t);
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.5, 0.5), wave_length * 0.7, steepness * 0.9, t);
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.1, 0.9), wave_length * 1.2, steepness * 1.1, t);
        // huge waves
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(1, 0), wave_length * 2, steepness * 1.5, t);
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.1, 0.9), wave_length * 1.9, steepness * 1.2, t);
        position += trochoidalWaveOffset(d_particles[ti.thread_id].relative, vec2(0.9, 0.1), wave_length * 2.1, steepness * 1.1, t);

        d_particles[ti.thread_id].position = position;
        d_particles[ti.thread_id].predicted = position;
    }
}

void ParticleSystem::update(float dt) {
    last_dt = dt;
    if (physics_mode == 0) {      
        const unsigned int BLOCKS_CELLS = Saiga::CUDA::getBlockCount(cellCount, BLOCK_SIZE);

        resetConstraintCounter<<<1, 32>>>(d_constraintCounter, d_constraintCounterWalls);
        resetCellListOptimized<<<BLOCKS_CELLS, BLOCK_SIZE>>>(d_cell_list, cellCount, particleCount);
        updateParticlesPBD1_radius<<<BLOCKS, BLOCK_SIZE>>>(dt, gravity, d_particles, damp_v, particleRadiusWater, particleRadiusCloth);
        
        calculateHash<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particle_hash, d_cell_list, d_particle_list, cellDim, cellCount, cellSize);
        thrust::sort_by_key(thrust::device_pointer_cast(d_particle_hash), thrust::device_pointer_cast(d_particle_hash) + particleCount, d_particles.device_begin());
        createLinkedCellsOptimized<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particle_hash, d_cell_list, d_particle_list, cellDim, cellCount, cellSize);
        
        createConstraintParticlesLinkedCellsRigidBodiesFluid<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize);
        createConstraintWalls<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_walls, d_constraintListWalls, d_constraintCounterWalls, maxConstraintNumWalls);
        
        updateLookupTable<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_particleIdLookup);
        CUDA_SYNC_CHECK_ERROR();

        float w_poly_d_q = W_poly6(delta_q * h, h);
        float calculatedRelaxP = relax_p;

        for (int i = 0; i < solver_iterations; i++) {
            computeDensityAndLambda<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, omega_lambda_relax, particle_radius_rest_density);
            updateParticlesPBD2IteratorFluid<<<BLOCKS, BLOCK_SIZE>>>(d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, particle_radius_rest_density, artificial_pressure_k, artificial_pressure_n, w_poly_d_q);

            if (use_calculated_relax_p) {
                calculatedRelaxP = 1 - pow(1 - calculatedRelaxP, 1.0/(i+1));
            }
            updateRigidBodies();

            solverPBDParticlesSDF<<<Saiga::CUDA::getBlockCount(maxConstraintNum, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintList, d_constraintCounter, maxConstraintNum, relax_p, d_rigidBodies, mu_k, mu_s, mu_f);
            solverPBDWalls<<<Saiga::CUDA::getBlockCount(maxConstraintNumWalls, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_walls, d_constraintListWalls, d_constraintCounterWalls, maxConstraintNumWalls, relax_p, mu_k, mu_s, mu_f);
            
            solverPBDCloth<<<Saiga::CUDA::getBlockCount(maxConstraintNumCloth, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintListCloth, d_constraintCounterCloth, maxConstraintNumCloth, d_particleIdLookup);
            if (test_bool)
                solverPBDClothBending<<<Saiga::CUDA::getBlockCount(maxConstraintNumClothBending, BLOCK_SIZE), BLOCK_SIZE>>>(d_particles, d_constraintListClothBending, d_constraintCounterClothBending, maxConstraintNumClothBending, d_particleIdLookup, test_float);
            
            updateParticlesPBD2Iterator<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, calculatedRelaxP);
            CUDA_SYNC_CHECK_ERROR();
        }

        //constraintsShapeMatchingRB
        //constraintsShapeMatchingRB();

        updateRigidBodies();
        controlRigidBody(0, control_forward, control_rotate, dt);
        resolveRigidBodyConstraints<<<BLOCKS, BLOCK_SIZE>>>(d_particles, particleCount, d_rigidBodies);
        CUDA_SYNC_CHECK_ERROR();

        updateParticlesPBD2<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, relax_p);
        updateTrochoidalParticles<<<BLOCKS, BLOCK_SIZE>>>(d_particles, wave_number, phase_speed, steepness, dt * steps);

        computeVorticityAndViscosity<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, c_viscosity);
        applyVorticityAndViscosity<<<BLOCKS, BLOCK_SIZE>>>(dt, d_particles, d_cell_list, d_particle_list, d_constraintList, d_constraintCounter, maxConstraintNum, cellDim, cellCount, cellSize, h, epsilon_spiky, epsilon_vorticity, wind_direction, wind_speed);
        CUDA_SYNC_CHECK_ERROR();
        
        hipDeviceSynchronize();
    }
    steps += 1;
}

__device__ float stabilize(float rot, float center, float stabilize, float max) {
    float new_rot = 0;
    if (rot > M_PI/2.0) {
        center += M_PI;
    } else if (rot < -M_PI/2.0) {
        center -= M_PI;
    }
    new_rot = rot * (1 - stabilize) + center * stabilize;
    return new_rot - rot;
}

__device__ float normalizeRotation(float rot) {
    if (rot > M_PI)
        rot = rot - 2 * M_PI;
    if (rot < -M_PI)
        rot = rot + 2 * M_PI;
    return rot;
}

__device__ void normalizeRotation(vec3 rot) {
    rot.x() = normalizeRotation(rot.x());
    rot.y() = normalizeRotation(rot.y());
    rot.z() = normalizeRotation(rot.z());
}

__global__ void moveRigidBody(Saiga::ArrayView<Particle> particles, int particleCountRB, RigidBody *rigidBodies, int rbID, float forward, float rotate) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id > 0)
        return;
    vec3 rot = rigidBodies[rbID].A.eulerAngles(1, 0, 2);
    vec3 initRot = rigidBodies[rbID].initA.eulerAngles(1, 0, 2);
    vec3 relRot = rot - initRot;

    // another approach
    vec3 direction3d = {1, 0, 0};
    direction3d = rigidBodies[rbID].A * direction3d;

    normalizeRotation(relRot);

    rot.x() += rotate * 0.001;
    rot.y() += stabilize(relRot.y(), 0, 0.01, M_PI/4.0);
    rot.z() += stabilize(relRot.z(), 0, 0.01, M_PI/4.0);
    
    normalizeRotation(rot);

    mat3 rotMat;
    rotMat = Eigen::AngleAxisf(rot.x(), vec3::UnitY())
        * Eigen::AngleAxisf(rot.y(), vec3::UnitX())
        * Eigen::AngleAxisf(rot.z(), vec3::UnitZ());
    rigidBodies[rbID].A = rotMat;

    //vec3 direction = {cosf(rotY), 0, sinf(rotY)};
    vec3 direction = {direction3d.x(), 0, direction3d.z()};
    direction.normalize();
    rigidBodies[rbID].originOfMass += direction * forward * 0.001;
}

void ParticleSystem::controlRigidBody(int rbID, float forward, float rotate, float dt){
    moveRigidBody<<<1, 32>>>(d_particles, particleCountRB, d_rigidBodies, rbID, forward, rotate);
}

// 2.3 Ray
__global__ void rayList(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    Particle &particle = particles[ti.thread_id];
    vec3 z = ray.origin - particle.position;
    float dz = ray.direction.dot(z);
    float Q = (dz * dz) - z.dot(z) + particle.radius * particle.radius;

    if (Q > 0) {
        int idx = atomicAdd(rayHitCount, 1);
        list[idx].first = ti.thread_id;
        list[idx].second = -dz;
    }
}

__global__ void rayColor(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, vec4 color) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    if (ti.thread_id == 0)
        particles[list[min].first].color = color;
    list[ti.thread_id].second = 0;
}

__global__ void rayImpulse(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= 1000)
        return;
    if (ti.thread_id == 0)
        particles[list[min].first].velocity += ray.direction * 42;
    list[ti.thread_id].second = 0;
}

__global__ void rayExplosion(Saiga::ArrayView<Particle> particles, Saiga::Ray ray, thrust::pair<int, float> *list, int *rayHitCount, int min, bool explode, float explodeMult) {
    Saiga::CUDA::ThreadInfo<> ti;
    if (ti.thread_id >= particles.size())
        return;
    if (ti.thread_id == list[min].first)
        return;
    Particle &explodeParticle = particles[list[min].first];
    Particle &particle = particles[ti.thread_id];
    vec3 dir = particle.position - explodeParticle.position;
    float d = dir.norm();
    if (!explode)
        explodeMult = -explodeMult;
    if (d < 4) {
        particle.velocity += 1.0 / (d) * dir * explodeMult;
        particle.d_momentum += 1.0 / (d) * dir * explodeMult;
    }
    list[ti.thread_id].second = 0;
}

// remove if
struct remove_predicate
{
  __host__ __device__
  bool operator()(const thrust::pair<int, float> x)
  {
    return x.second <= 1e-5;
  }
};
// min element
struct compare_predicate
{
  __host__ __device__
  bool operator()(thrust::pair<int, float> a, thrust::pair<int, float> b)
  {
    return a.second < b.second;
  }
};

void ParticleSystem::ray(Saiga::Ray ray) {
    thrust::device_vector<thrust::pair<int, float>> d_vec(1000);
    resetCounter<<<1, 32>>>(d_rayHitCount);
    rayList<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount);
    CUDA_SYNC_CHECK_ERROR();
    int N = thrust::remove_if(d_vec.begin(), d_vec.end(), remove_predicate()) - d_vec.begin();
    if (N == 0)
        return;
    int min = thrust::min_element(d_vec.begin(), d_vec.begin() + N, compare_predicate()) - d_vec.begin();
    if (action_mode == 0) {
        rayColor<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, color);
    } else if (action_mode == 1) {
        rayImpulse<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min);
    } else if (action_mode == 2) {
        rayExplosion<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, true, explosion_force);
    } else if (action_mode == 3) {
        rayExplosion<<<BLOCKS, BLOCK_SIZE>>>(d_particles, ray, thrust::raw_pointer_cast(&d_vec[0]), d_rayHitCount, min, false, explosion_force);
    }
    CUDA_SYNC_CHECK_ERROR();
}