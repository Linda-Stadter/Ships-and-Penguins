#include "hip/hip_runtime.h"
﻿#include "particleSystem.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/core/util/assert.h"

#include "saiga/core/math/Eigen_Compile_Checker.h"

__host__ void checkError(hipError_t err);

ParticleSystem::ParticleSystem(int _particleCount, vec3 _boxMin, vec3 _boxDim)
    : particleCount(_particleCount), boxMin(_boxMin), boxDim(_boxDim)
{
    hipStreamCreate(&stream1); hipStreamCreate(&stream2); hipStreamCreate(&stream3);

    checkError(hipMalloc((void **)&d_constraintList, sizeof(int) * maxConstraintNum * 2));
    checkError(hipMalloc((void **)&d_constraintCounter, sizeof(int)));
    checkError(hipMalloc((void **)&d_constraintListWalls, sizeof(int) * maxConstraintNumWalls * 2));
    checkError(hipMalloc((void **)&d_constraintCounterWalls, sizeof(int)));
    checkError(hipMalloc((void **)&d_rayHitCount, sizeof(int)));

    checkError(hipMalloc((void **)&d_constraintListCloth, sizeof(ClothConstraint) * maxConstraintNumCloth));
    checkError(hipMalloc((void **)&d_constraintCounterCloth, sizeof(int)));
    checkError(hipMalloc((void **)&d_constraintListClothBending, sizeof(ClothBendingConstraint) * maxConstraintNumClothBending));
    checkError(hipMalloc((void **)&d_constraintCounterClothBending, sizeof(int)));

    checkError(hipMalloc((void **)&d_particleIdLookup, sizeof(int) * particleCount));
    
    checkError(hipMalloc((void **)&d_rigidBodies, sizeof(RigidBody) * maxRigidBodyCount));

    float minCellSize = 2.0 * maxParticleRadius;
    cellSize = minCellSize;
    cellDim = {int(ceil(boxDim[0] / cellSize)), int(ceil(boxDim[1] / cellSize)), int(ceil(boxDim[2] / cellSize))};
    cellCount = cellDim[0] * cellDim[1] * cellDim[2];

    checkError(hipMalloc((void**)&d_particle_list, sizeof(int) * particleCount));
	checkError(hipMalloc((void**)&d_cell_list, sizeof(std::pair<int, int>) * cellCount));
	checkError(hipMalloc((void**)&d_particle_hash, sizeof(int) * particleCount));

    std::cout << "ParticleSystem initialized!" << std::endl;
    CUDA_SYNC_CHECK_ERROR();
}

ParticleSystem::~ParticleSystem()
{
    checkError(hipFree(d_constraintList));
    checkError(hipFree(d_constraintCounter));
    checkError(hipFree(d_constraintListWalls));
    checkError(hipFree(d_constraintCounterWalls));
    checkError(hipFree(d_rayHitCount));

    checkError(hipFree(d_particle_list));
	checkError(hipFree(d_cell_list));

	checkError(hipFree(d_rigidBodies));

    std::cout << "~ParticleSystem" << std::endl;
}
