#include "hip/hip_runtime.h"
﻿#include "particleSystem.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/core/util/assert.h"

#include "saiga/core/math/Eigen_Compile_Checker.h"

__host__ void checkError(hipError_t err);

ParticleSystem::ParticleSystem(int _particleCount, vec3 _boxMin, vec3 _boxDim)
    : particleCount(_particleCount), boxMin(_boxMin), boxDim(_boxDim)
{
    hipStreamCreate(&stream1); hipStreamCreate(&stream2); hipStreamCreate(&stream3);

    checkError(hipMalloc((void **)&d_constraintList, sizeof(int) * maxConstraintNum*2));
    checkError(hipMalloc((void **)&d_constraintCounter, sizeof(int) * 1));
    checkError(hipMalloc((void **)&d_constraintListWalls, sizeof(int) * maxConstraintNumWalls*2));
    checkError(hipMalloc((void **)&d_constraintCounterWalls, sizeof(int) * 1));
    checkError(hipMalloc((void **)&d_rayHitCount, sizeof(int) * 1));

    float minCellSize = 2.0 * maxParticleRadius;
    cellSize = minCellSize;
    cellDim = {int(ceil(boxDim[0] / cellSize)), int(ceil(boxDim[1] / cellSize)), int(ceil(boxDim[2] / cellSize))};
    cellCount = cellDim[0] * cellDim[1] * cellDim[2];

    checkError(hipMalloc((void**)&d_particle_list, sizeof(int) * particleCount));
	checkError(hipMalloc((void**)&d_cell_list, sizeof(int) * cellCount));

    std::cout << "ParticleSystem initialized!" << std::endl;
    CUDA_SYNC_CHECK_ERROR();
}

ParticleSystem::~ParticleSystem()
{
    checkError(hipFree(d_constraintList));
    checkError(hipFree(d_constraintCounter));
    checkError(hipFree(d_constraintListWalls));
    checkError(hipFree(d_constraintCounterWalls));
    checkError(hipFree(d_rayHitCount));

    checkError(hipFree(d_particle_list));
	checkError(hipFree(d_cell_list));

    std::cout << "~ParticleSystem" << std::endl;
}
