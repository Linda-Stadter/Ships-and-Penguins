﻿#include "particleSystem.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/core/util/assert.h"

#include "saiga/core/math/Eigen_Compile_Checker.h"

__host__ void checkError(hipError_t err);

ParticleSystem::ParticleSystem(int _particleCount)
    : particleCount(_particleCount)
{
    checkError(hipMalloc((void **)&d_constraintList, sizeof (int) * maxConstraintNum*2));
    checkError(hipMalloc((void **)&d_constraintCounter, sizeof (int) * 1));
    checkError(hipMalloc((void **)&d_constraintListWalls, sizeof (int) * maxConstraintNumWalls*2));
    checkError(hipMalloc((void **)&d_constraintCounterWalls, sizeof (int) * 1));
    checkError(hipMalloc((void **)&d_rayHitCount, sizeof (int) * 1));
    std::cout << "ParticleSystem initialized!" << std::endl;
    CUDA_SYNC_CHECK_ERROR();
}

ParticleSystem::~ParticleSystem()
{
    checkError(hipFree(d_constraintList));
    checkError(hipFree(d_constraintCounter));
    checkError(hipFree(d_constraintListWalls));
    checkError(hipFree(d_constraintCounterWalls));
    checkError(hipFree(d_rayHitCount));
    std::cout << "~ParticleSystem" << std::endl;
}
