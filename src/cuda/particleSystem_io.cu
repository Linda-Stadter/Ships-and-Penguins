﻿#include "saiga/core/imgui/imgui.h"
#include "saiga/core/math/random.h"
#include "saiga/core/util/assert.h"

#include "particleSystem.h"

#include <thrust/extrema.h>
#include <thrust/sort.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

void ParticleSystem::renderGUI()
{

    if(ImGui::Begin("ParticleSystem"))
    {
        ImGui::InputFloat3("gravity", &gravity[0]);
        ImGui::InputFloat("elast", &elast_const);
        ImGui::InputFloat("spring", &spring_const);
        ImGui::InputFloat("frict", &frict_const);

        ImGui::InputInt("solverIterations", &solverIterations);
        ImGui::Checkbox("useCalculatedRelaxP", &useCalculatedRelaxP);
        ImGui::InputFloat("relax", &relaxP);
        ImGui::InputFloat("damp", &dampV);
        ImGui::Checkbox("Jacobi Solver (or Gauss-Seidel)", &jacobi);

        ImGui::InputFloat("particleRadiusRestDensity", &particleRadiusRestDensity);
        ImGui::InputFloat("particleRadiusWater", &particleRadiusWater);
        ImGui::InputFloat("particleRadiusCloth", &particleRadiusCloth);

        ImGui::Checkbox("test bool", &testBool);
        ImGui::InputFloat("test float", &testFloat);

        ImGui::InputFloat("h", &h);
        ImGui::InputFloat("epsilon_spiky", &epsilon_spiky);
        ImGui::InputFloat("omega_lambda_relax", &omega_lambda_relax);

        ImGui::InputFloat("artificial_pressure_k", &artificial_pressure_k);
        ImGui::InputInt("artificial_pressure_n", &artificial_pressure_n);
        ImGui::InputFloat("delta_q", &delta_q);

        ImGui::InputFloat("c_viscosity", &c_viscosity);
        ImGui::InputFloat("epsilon_vorticity", &epsilon_vorticity);

        ImGui::Combo("physics", &physicsMode, physics, std::size(physics));
        ImGui::Checkbox("use SDF", &useSDF); // 4.4
        ImGui::Combo("mouse action", &actionMode, actions, std::size(actions));
        ImGui::ColorEdit4("color", &color[0]);
        ImGui::InputInt("explosionForce", &explosionForce);
        ImGui::InputInt("splitCount", &splitCount);

        ImGui::Separator();
        ImGui::Combo("hashing", &hashFunction, hashes, std::size(hashes));
    }
    ImGui::End();
}
