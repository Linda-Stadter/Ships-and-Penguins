﻿#include "saiga/core/imgui/imgui.h"
#include "saiga/core/math/random.h"
#include "saiga/core/util/assert.h"

#include "particleSystem.h"

#include <thrust/extrema.h>
#include <thrust/sort.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

void ParticleSystem::renderGUI()
{

    if(ImGui::Begin("ParticleSystem"))
    {
        ImGui::InputFloat3("gravity", &gravity[0]);

        ImGui::InputInt("solver_iterations", &solver_iterations);
        ImGui::Checkbox("use_calculated_relax_p", &use_calculated_relax_p);
        ImGui::InputFloat("relax", &relax_p);
        ImGui::InputFloat("damp", &damp_v);

        ImGui::InputFloat("particle_radius_rest_density", &particle_radius_rest_density);
        ImGui::InputFloat("particleRadiusWater", &particleRadiusWater);
        ImGui::InputFloat("particleRadiusCloth", &particleRadiusCloth);

        ImGui::Checkbox("test bool", &test_bool);
        ImGui::InputFloat("test float", &test_float);

        ImGui::InputFloat("cloth_break_distance", &cloth_break_distance);

        ImGui::InputFloat("kinetic friction", &mu_k);
        ImGui::InputFloat("static friction", &mu_s);
        ImGui::InputFloat("friction", &mu_f);

        ImGui::InputFloat("h", &h);
        ImGui::InputFloat("epsilon_spiky", &epsilon_spiky);
        ImGui::InputFloat("omega_lambda_relax", &omega_lambda_relax);

        ImGui::InputFloat("artificial_pressure_k", &artificial_pressure_k);
        ImGui::InputInt("artificial_pressure_n", &artificial_pressure_n);
        ImGui::InputFloat("delta_q", &delta_q);

        ImGui::InputFloat("c_viscosity", &c_viscosity);
        ImGui::InputFloat("epsilon_vorticity", &epsilon_vorticity);

        ImGui::InputFloat3("cannonball_speed", &cannonball_speed[0]);

        ImGui::InputFloat("wave_length", &wave_number);
        ImGui::InputFloat("phase_speed", &phase_speed);
        ImGui::InputFloat("steepness", &steepness);

        ImGui::InputFloat3("wind_direction", &wind_direction[0]);
        ImGui::InputFloat("wind_speed", &wind_speed);

        ImGui::Combo("physics", &physics_mode, physics, std::size(physics));
        ImGui::Combo("mouse action", &action_mode, actions, std::size(actions));
        ImGui::ColorEdit4("color", &color[0]);
        ImGui::InputInt("explosion_force", &explosion_force);
        ImGui::InputInt("split_count", &split_count);

        ImGui::Separator();
    }
    ImGui::End();
}

void ParticleSystem::renderIngameGUI()
{
    // game ui
    ImGui::SetNextWindowPos(ImVec2(0, 0), ImGuiCond_Always);
    ImGui::SetNextWindowSize(ImVec2(200, 900), ImGuiCond_Always);
    if(ImGui::Begin("TEST", NULL, ImGuiWindowFlags_NoResize | ImGuiWindowFlags_NoScrollbar | ImGuiWindowFlags_NoTitleBar | ImGuiWindowFlags_NoBackground))
    {
        ImGui::TextColored(ImColor(1.f, 1.f, 1.f, 1.f), "Time: ");
        int reload_bar_length = 10;
        int reload_bar_progress = (float)cannon_timer / (float)cannon_timer_reset * reload_bar_length;
        std::string reload = "reload: ";
        for (int i = 0; i < reload_bar_progress; i++) {
            reload.append("#");
        }
        ImColor reload_color = ImColor(1.f, 1.f, 1.f, 1.f);
        if (reload_bar_progress == reload_bar_length)
            reload_color = ImColor(.5f, 1.f, .5f, 1.f);
        ImGui::TextColored(reload_color, reload.c_str());
    }
    ImGui::End();
}
